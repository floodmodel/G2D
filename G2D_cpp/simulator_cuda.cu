#include "hip/hip_runtime.h"
﻿#ifdef __INTELLISENSE___
void __syncthreads();
#else
void __syncthreads();
#endif

#include "stdafx.h"
#include <ATLComTime.h>

#include "g2d.h"
#include "g2d_cuda.cuh"

using namespace std;
namespace fs = std::filesystem;

extern fs::path fpn_log;

extern projectFile prj;
extern domaininfo di;
extern domainCell** dmcells;
extern generalEnv ge;

extern cvatt* cvs;
extern cvattAddAtt* cvsAA;
extern double* cvsele;
extern double* rfi_read_mPs;
extern bcAppinfo * bcAppinfos;

extern thisProcess ps;
extern thisProcessInner psi;
extern globalVinner gvi;
extern dataForCalDT dataForDT;
extern minMaxFlux mnMxFluxFromAllcells;


//#ifdef OnGPU
int simulationControl_GPU()
{
	hipDeviceProp_t m_deviceProp;
	hipGetDeviceProperties(&m_deviceProp, 0);
	if (!m_deviceProp.canMapHostMemory) {
		fprintf(stderr, "Device does not support mapping CPU host memory!\n");
		return 0;
	}
	double simDur_min = 0.0;
	if (prj.printOutInterval_min > 1) {
		simDur_min = prj.simDuration_min + 1.0;
	}
	else {
		simDur_min = prj.simDuration_min;
	}
	int bcDataOrder = 0;
	int rfDataOrder = 0;
	int demToChangeEnded = 1;
	double tnow_min_bak = 0.0;
	int dtbc_min = prj.bcDataInterval_min;
	if (prj.isDEMtoChangeApplied == 1) {
		demToChangeEnded = 0;
	}
	initThisProcess();
	initGlobalVinner();
	initFloodingThresholds();
	initMinMax();
	cvatt* d_cvs;
	cvattAddAtt* d_cvsAA;
	double* d_cvsele;
	double* d_rfi_read_mPs;
	double* d_dtsec;
	bcAppinfo* d_bcAppinfos;
	minMaxFlux* d_minMaxCVidx;

	size_t  ms_cvs_ncvs = di.cellNnotNull * sizeof(cvatt);
	size_t  ms_cvsAA_ncvs = di.cellNnotNull * sizeof(cvattAddAtt);
	size_t ms_double_ncvs = di.cellNnotNull * sizeof(double);
	size_t ms_bcAppinfo_allBCcells = prj.bcCellCountAll * sizeof(bcAppinfo);
	size_t ms_minMaxCVidx = sizeof(minMaxFlux);
	size_t ms_minMaxCVidx_TPB = ps.threadsPerBlock * sizeof(minMaxFlux);

	dim3 thPblock(ps.threadsPerBlock, 1, 1); //blockDim    
	dim3 bPgrid(di.cellNnotNull / (thPblock.x * thPblock.y) + 1, 1); //gridDim 

	//clock_t  ts, tf;
	//long tc_memcopy = 0;
	//long tc_setStartingConditionCVs_GPU = 0;
	//long tc_set_rf_bc_dem = 0;
	//long tc_initilizeThisStep_GPU = 0;
	//long tc_GSiteration = 0;
	//long tc_getMinMaxFromCV = 0;
	//long tc_getMinMax_reduction = 0;
	//long tc_memcpy_mnMxCVidx = 0;
	//long tc_setAllCVFalse = 0;

	//ts = clock();
	hipMalloc((void**)& d_cvs, ms_cvs_ncvs);
	hipMalloc((void**)& d_cvsAA, ms_cvsAA_ncvs);
	hipMalloc((void**)& d_cvsele, ms_double_ncvs);
	hipMalloc((void**)& d_bcAppinfos, ms_bcAppinfo_allBCcells);
	hipMalloc((void**)& d_rfi_read_mPs, ms_double_ncvs);
	hipMalloc((void**)& d_minMaxCVidx, bPgrid.x * sizeof(minMaxFlux));
	hipMalloc((void**)& d_dtsec, sizeof(double));
	hipMemcpy(d_cvs, cvs, ms_cvs_ncvs, hipMemcpyHostToDevice);
	hipMemcpy(d_cvsAA, cvsAA, ms_cvsAA_ncvs, hipMemcpyHostToDevice);
	hipMemcpy(d_cvsele, cvsele, ms_double_ncvs, hipMemcpyHostToDevice);
	hipMemcpy(d_bcAppinfos, bcAppinfos, ms_bcAppinfo_allBCcells, hipMemcpyHostToDevice);
	hipMemcpy(d_rfi_read_mPs, rfi_read_mPs, ms_double_ncvs, hipMemcpyHostToDevice);
	//tf = clock();
	//tc_memcopy = long(tf - ts);

	//ts = clock();
	//setStartingConditionCVs_GPU << < bPgrid, thPblock >> > (d_cvs, d_cvsAA, d_cvsele, gvi.nCellsInnerDomain);
	setStartingConditionCVs_GPU << < bPgrid, thPblock >> > (d_cvs, d_cvsAA, d_cvsele, gvi.nCellsInnerDomain);
	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());
	//tf = clock();
	//tc_setStartingConditionCVs_GPU = long(tf - ts);

	do { //모의 시작할 때 t 는 초기 조건, t+dt는 소스 하나가 적용된 결과
		psi.tnow_min = psi.tnow_sec / 60.0;
		//ts = clock();
		if (prj.isbcApplied == 1) {//경계조건 등
			int bc_min = bcDataOrder * dtbc_min;
			if (((tnow_min_bak < bc_min) & (psi.tnow_min >= bc_min))
				|| bc_min == 0) {
				bcDataOrder++;
				getCellCD(bcDataOrder, dtbc_min); // 경계조건 값은 포인터로 넘긴다..
				hipMemcpy(d_bcAppinfos, bcAppinfos, ms_bcAppinfo_allBCcells, hipMemcpyHostToDevice);
			}
		}
		if (prj.isRainfallApplied == 1 && psi.rfEnded == 0) {//강우
			int rf_min = rfDataOrder * prj.rainfallDataInterval_min;
			if (((tnow_min_bak < rf_min) & (psi.tnow_min >= rf_min))
				|| rf_min == 0) {
				rfDataOrder++; //1부터 시작. 배열은 rainfallDataOrder-1
				psi.rfEnded = readRainfallAndGetIntensity(rfDataOrder);
				hipMemcpy(d_rfi_read_mPs, rfi_read_mPs, ms_double_ncvs, hipMemcpyHostToDevice);
			}
		}
		if (prj.isDEMtoChangeApplied == 1 && demToChangeEnded == 0) {//dem file 교체
			demToChangeEnded = changeDomainElevWithDEMFile(psi.tnow_min, tnow_min_bak);
			hipMemcpy(d_cvsele, cvsele, ms_double_ncvs, hipMemcpyHostToDevice);
		}
		//CUDA_CHECK(hipDeviceSynchronize());
		//tf = clock();
		//tc_set_rf_bc_dem = long(tf - ts);

		//ts = clock();
		initilizeThisStep_GPU << < bPgrid, thPblock >> > (d_cvs, d_cvsAA, d_cvsele, d_bcAppinfos, d_rfi_read_mPs, psi, gvi);
		CUDA_CHECK(hipGetLastError());
		CUDA_CHECK(hipDeviceSynchronize());
		//tf = clock();
		//tc_initilizeThisStep_GPU = long(tf - ts);

		//ts = clock();
		runSolver_GPU << < bPgrid, thPblock >> > (d_cvs, d_bcAppinfos, d_cvsele, gvi);
		CUDA_CHECK(hipGetLastError());
		CUDA_CHECK(hipDeviceSynchronize());
		//tf = clock();
		//tc_GSiteration = long(tf - ts);

		//ts = clock();
		//getMinMaxFromCV << < bPgrid, thPblock, ms_minMaxCVidx_TPB >> > (d_cvs,	d_cvsAA, gvi, d_minMaxCVidx);
		updateGlobalMinMaxFromCV << < bPgrid, thPblock, ms_minMaxCVidx_TPB >> > (d_cvs, gvi, d_minMaxCVidx);
		CUDA_CHECK(hipGetLastError());
		CUDA_CHECK(hipDeviceSynchronize());
		//tf = clock();
		//tc_getMinMaxFromCV = long(tf - ts);

		// reduction=================================
		//ts = clock();
		if (bPgrid.x > 1) {
			int array_size = bPgrid.x;
			int numBlock = (array_size + thPblock.x - 1) / thPblock.x + 1;;
			while (numBlock != 1) {
				if (array_size < ps.threadsPerBlock) {
					numBlock = 1;
					updateGlobalMinMaxFromArray << < numBlock, thPblock, ms_minMaxCVidx_TPB >> > (d_minMaxCVidx,
						array_size, gvi.isApplyVNC, d_minMaxCVidx);
					//CUDA_CHECK(hipGetLastError());
					CUDA_CHECK(hipDeviceSynchronize()); // 커널함수 들이 완료될때 까지 대기, block 동기화
					break;
				}
				else {
					updateGlobalMinMaxFromArray << < numBlock, thPblock, ms_minMaxCVidx_TPB >> > (d_minMaxCVidx,
						array_size, gvi.isApplyVNC, d_minMaxCVidx);
					//CUDA_CHECK(hipGetLastError());
					CUDA_CHECK(hipDeviceSynchronize());// 커널함수 들이 완료될때 까지 대기, block 동기화 
					array_size = numBlock;
					numBlock = (numBlock + thPblock.x - 1) / thPblock.x + 1;
					
				}
			}
		}
		//tf = clock();
		//tc_getMinMax_reduction = long(tf - ts);
		//==========================================
		if (psi.tnow_sec >= psi.tsec_targetToprint) {
			hipMemcpy(cvs, d_cvs, ms_cvs_ncvs, hipMemcpyDeviceToHost);
			updateSummaryAndSetAllFalse();// 출력할때 마다 이 정보 업데이트
			//updateSummaryAndSetAllFalse_serial(); // openMP 병렬계산과 결과 같음
			//ts = clock();
			setAllCVFalse << <bPgrid, thPblock >> > (d_cvs, gvi.nCellsInnerDomain); // cvs.isSimulatingCell 을 복사하지 않고, 여기서 d_cvs에서 설정해 준다.
			CUDA_CHECK(hipGetLastError());
			CUDA_CHECK(hipDeviceSynchronize());
			//tf = clock();
			//tc_setAllCVFalse = long(tf - ts);
			//cout << "\n\nCurrent min           : " << psi.tnow_min << "min" << endl;
			//cout << "Time consumed malloc, memcopy          : " << tc_memcopy << "ms" << endl;
			//cout << "Time consumed setStartingCondition_GPU : " << tc_setStartingConditionCVs_GPU << "ms" << endl;
			//cout << "Time consumed tc_set_rf_bc_dem         : " << tc_set_rf_bc_dem << "ms" << endl;
			//cout << "Time consumed tc_initilizeThisStep_GPU : " << tc_initilizeThisStep_GPU << "ms" << endl;
			//cout << "Time consumed tc_GSiteration           : " << tc_GSiteration << "ms" << endl;
			//cout << "Time consumed tc_getMinMaxFromCV       : " << tc_getMinMaxFromCV << "ms" << endl;
			//cout << "Time consumed tc_getMinMax_reduction   : " << tc_getMinMax_reduction << "ms" << endl;
			//cout << "Time consumed tc_memcpy_mnMxCVidx      : " << tc_memcpy_mnMxCVidx << "ms" << endl;
			//cout << "Time consumed tc_setAllCVFalse         : " << tc_setAllCVFalse << "ms" << endl;
			makeOutputFiles(psi.tnow_sec, gvi.iGSmaxLimit);
			int progressRatio = (int)(psi.tnow_min / prj.simDuration_min * 100);
			printf("\rCurrent progress[min]: %d/%d[%d%%]..", (int)psi.tnow_min,
				(int)prj.simDuration_min, progressRatio);
			//한번 출력할때 마다 모의변수 업데이트
			if (updateProjectParameters() == 0) { return 0; }
			else if (prj.parChanged == 1) {
				initThisProcess();
				initGlobalVinner();
				initFloodingThresholds();
				if (prj.printOutInterval_min > 1) {
					simDur_min = prj.simDuration_min + 1.0;
				}
				else {
					simDur_min = prj.simDuration_min;
				}
			}
			psi.tsec_targetToprint = psi.tsec_targetToprint + ps.dt_printout_sec;
			ps.thisPrintStepStartTime = COleDateTime::GetCurrentTime();
		}
		tnow_min_bak = psi.tnow_min;
		psi.tnow_sec = psi.tnow_sec + gvi.dt_sec;
		if (prj.isFixedDT == 0) {
			//ts = clock();
			hipMemcpy(&mnMxFluxFromAllcells, d_minMaxCVidx, ms_minMaxCVidx, hipMemcpyDeviceToHost); // dt를 계산하려면, 매번 받아와야 한다. 
			// bcAppinfos_L[bci].bcDepth_dt_m_tp1를 initializeThisStepAcell()에서 매번 계산하므로
			// dt를 계산하려면, 매번 받아와야 한다.
			hipMemcpy(bcAppinfos, d_bcAppinfos, ms_bcAppinfo_allBCcells, hipMemcpyDeviceToHost); 
			//tf = clock();
			//tc_memcpy_mnMxCVidx = long(tf - ts);
			gvi.dt_sec = getDTsecWithConstraints(dataForDT, gvi, psi.tnow_sec, bcAppinfos,
				mnMxFluxFromAllcells);
		}
	} while (psi.tnow_min < simDur_min);

	CUDA_CHECK(hipFree(d_cvs));
	CUDA_CHECK(hipFree(d_cvsAA));
	CUDA_CHECK(hipFree(d_cvsele));
	CUDA_CHECK(hipFree(d_bcAppinfos));
	CUDA_CHECK(hipFree(d_rfi_read_mPs));
	return 1;
}

__global__ void initilizeThisStep_GPU(cvatt* d_cvs, cvattAddAtt* d_cvsAA, double * d_cvsele,
	bcAppinfo* d_bcApp, double* d_rfi_read_mPs, 
	thisProcessInner psi_k, globalVinner gvi_k){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < gvi_k.nCellsInnerDomain) {
		initializeThisStepAcell(d_cvs, d_cvsAA, d_bcApp, d_cvsele[idx], d_rfi_read_mPs[idx], idx, psi_k, gvi_k);
		if (d_cvs[idx].dp_tp1 >dMinLimit) {
			setEffCells(d_cvs, idx);
		}
	}
}

__global__ void setStartingConditionCVs_GPU(cvatt* d_cvs, cvattAddAtt * d_cvsAA, 
	double* d_cvsele, int arraySize) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < arraySize) {
		setStartingConditionCVs_inner(d_cvs, d_cvsAA, d_cvsele, tid);
	}
}

__global__ void updateGlobalMinMaxFromCV(cvatt* cvs_k, //cvattAddAtt* cvsAA_k,
	globalVinner gvi_k, minMaxFlux* odata) {
	/*__shared__ minMaxCVidx sdata[THPB]; */
	extern __shared__ minMaxFlux sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	sdata[tid].dflowmaxInThisStep = -9999.0; // 여기에서 초기화 해준다. 실제 배열 길이가 < tid + s 인경우에도 값을 입력..
	sdata[tid].vmaxInThisStep = -9999.0;
	sdata[tid].VNConMinInThisStep = 9999.0;
	fluxNfd flxmax;
	if (idx < gvi_k.nCellsInnerDomain) {
		flxmax = get_maxFlux_FD(cvs_k, idx); // 셀별 max 값을 찾아서 global max 찾는데 이용
		sdata[tid].dflowmaxInThisStep = flxmax.dflow;
		sdata[tid].vmaxInThisStep = flxmax.v;
		if (gvi_k.isApplyVNC == 1) {
			sdata[tid].VNConMinInThisStep = getVNConditionValue(cvs_k, idx);
		}
	}
	__syncthreads(); // 초기화 부분에서 최대한 sync 시킨다. // 필수. 중요
	//for (int s = blockDim.x / 2; s > 0; s /= 2) {
	for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
		if (tid < s && idx < gvi_k.nCellsInnerDomain) {
			if (sdata[tid].dflowmaxInThisStep < sdata[tid + s].dflowmaxInThisStep) {
				sdata[tid].dflowmaxInThisStep = sdata[tid + s].dflowmaxInThisStep;
			}
			if (sdata[tid].vmaxInThisStep < sdata[tid + s].vmaxInThisStep) {
				sdata[tid].vmaxInThisStep = sdata[tid + s].vmaxInThisStep;
			}
			if (gvi_k.isApplyVNC == 1) {
				if (sdata[tid].VNConMinInThisStep > sdata[tid + s].VNConMinInThisStep) {
					sdata[tid].VNConMinInThisStep = sdata[tid + s].VNConMinInThisStep;
				}
			}
		}
		__syncthreads();
	}
	if (tid == 0) {
		odata[blockIdx.x].dflowmaxInThisStep = sdata[0].dflowmaxInThisStep;
		odata[blockIdx.x].vmaxInThisStep = sdata[0].vmaxInThisStep;
		odata[blockIdx.x].VNConMinInThisStep = sdata[0].VNConMinInThisStep;
	}
	//}
}
__global__ void updateGlobalMinMaxFromArray(minMaxFlux* minMaxCVidx_k, int arraySize,
	int applyVNC, minMaxFlux* odata) {
	extern  __shared__ minMaxFlux sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	sdata[tid].dflowmaxInThisStep = -9999.0; // 여기에서 초기화 해준다. 실제 배열 길이가 < tid + s 인경우에도 값을 입력..
	sdata[tid].vmaxInThisStep = -9999.0;
	sdata[tid].VNConMinInThisStep = 9999.0;
	if (idx < arraySize) {
		sdata[tid].dflowmaxInThisStep = minMaxCVidx_k[idx].dflowmaxInThisStep;
		sdata[tid].vmaxInThisStep = minMaxCVidx_k[idx].vmaxInThisStep;
		sdata[tid].VNConMinInThisStep = minMaxCVidx_k[idx].VNConMinInThisStep;
	}
	__syncthreads(); // 초기화 부분에서 최대한 sync 시킨다. // 필수. 중요
	//for (int s = blockDim.x / 2; s > 0; s /= 2) {
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s && idx < arraySize) {
			if (sdata[tid].dflowmaxInThisStep < sdata[tid + s].dflowmaxInThisStep) {
				sdata[tid].dflowmaxInThisStep = sdata[tid + s].dflowmaxInThisStep;
			}
			if (sdata[tid].vmaxInThisStep < sdata[tid + s].vmaxInThisStep) {
				sdata[tid].vmaxInThisStep = sdata[tid + s].vmaxInThisStep;
			}
			if (applyVNC == 1) {
				if (sdata[tid].VNConMinInThisStep > sdata[tid + s].VNConMinInThisStep) {
					sdata[tid].VNConMinInThisStep = sdata[tid + s].VNConMinInThisStep;
				}
			}
		}
		__syncthreads();
	}
	if (tid == 0) {
		odata[blockIdx.x].dflowmaxInThisStep = sdata[0].dflowmaxInThisStep;
		odata[blockIdx.x].vmaxInThisStep = sdata[0].vmaxInThisStep;
		odata[blockIdx.x].VNConMinInThisStep = sdata[0].VNConMinInThisStep;
	}
}


__global__ void setAllCVFalse(cvatt* d_cvs, int arraySize) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < arraySize) {
		d_cvs[tid].isSimulatingCell = 0;
	}
}


// 2021.05.28. 안정적인 __syncthreads() 를 위해서 전용으로 만듬.
__global__ void runSolver_GPU(cvatt* cvs_k, bcAppinfo* bcAppinfos_k,	double* cvsele_k, globalVinner gvi_k) {
	int nCells = gvi_k.nCellsInnerDomain;
	int igsLimit = gvi_k.iGSmaxLimit;
	int iNRLimit = gvi_k.iNRmaxLimit;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	double bcdepth = 0.0;
	int applyBCdepth = 0; // 1 : true, 0 : false
	double c1_IM = gvi_k.dt_sec / gvi_k.dx;


	if (idx < nCells && cvs_k[idx].isBCcell == 1) {
		int bcidx = getBcAppinfoidx(bcAppinfos_k, gvi_k.bcCellCountAll, idx);
		if (bcAppinfos_k[bcidx].bctype == 2 || bcAppinfos_k[bcidx].bctype == 3) {// 1:Discharge, 2:Depth, 3:WaterLevel, 4:None
			bcdepth = bcAppinfos_k[bcidx].bcDepth_dt_m_tp1;
			applyBCdepth = 1;
		}
	}
	for (int igs = 0; igs < igsLimit; ++igs) {
		int continueNR_aCell = 1;
		double dp_old = 0.0;
		double dn = 0.0;
		if (idx < nCells && cvs_k[idx].isSimulatingCell == 1) {
			dp_old = cvs_k[idx].dp_tp1;
		}
		for (int inr = 0; inr < iNRLimit; ++inr) {
			int calNR = -1;
			if (idx < nCells && cvs_k[idx].isSimulatingCell == 1 && continueNR_aCell == 1) {
				calNR = 1;
				dn = cvs_k[idx].dp_tp1;
			}
			if (calNR == 1) { calWFlux(cvs_k, cvsele_k, gvi_k, idx); }
			__syncthreads(); //sync_01 // if 문 밖에서 써야 한다. if 문 안에서 사용하면, 무한루프 된다.
			if (calNR == 1) { calEFlux(cvs_k, cvsele_k, gvi_k, idx); }
			if (calNR == 1) { calNFlux(cvs_k, cvsele_k, gvi_k, idx);	}
			__syncthreads(); //sync_02
			if (calNR == 1) { calSFlux(cvs_k, cvsele_k, gvi_k, idx); }
			__syncthreads(); //sync_03 
			if (calNR == 1) { 
				// 현재 셀의 수위가 올라가려면  -> qe-, qw+, qs-, qn+
				double fn = dn - cvs_k[idx].dp_t + (cvs_k[idx].qe_tp1 - cvs_k[idx].qw_tp1
					+ cvs_k[idx].qs_tp1 - cvs_k[idx].qn_tp1) * c1_IM;//- sourceTerm; 
				double eElem = pow(cvs_k[idx].dfe, 2 / 3.0) * sqrt(abs(cvs_k[idx].slpe)) / cvs_k[idx].rc;
				double sElem = pow(cvs_k[idx].dfs, 2 / 3.0) * sqrt(abs(cvs_k[idx].slps)) / cvs_k[idx].rc;
				double dfn = 1 + (eElem + sElem) * (5.0 / 3.0) * c1_IM;
				if (dfn == 0.0) {
					continueNR_aCell = -1;
				}
				else {
					double dnp1 = 0.0;
					if (applyBCdepth == 1) {
						dnp1 = bcdepth;
					}
					else {
						dnp1 = dn - fn / dfn;
					}
					if (dnp1 < 0.0) { dnp1 = 0.0; }
					//double resd =;
					cvs_k[idx].dp_tp1 = dnp1;
					cvs_k[idx].hp_tp1 = cvs_k[idx].dp_tp1 + cvsele_k[idx];
					if (abs(dnp1 - dn) <= CCh) {
						continueNR_aCell = -1;
					}
				}
			}
			__syncthreads(); //sync_04
		}
		//__syncthreads(); //sync_04_01
		if (idx < nCells) {
			cvs_k[idx].resd = abs(cvs_k[idx].dp_tp1 - dp_old);
			////2021.08.04 // 2021.08.05 NR 밖에서 이것을 이용하면, 각 함수에서 v 계산하는 것 보다 느려진다.
			//cvs_k[idx].ve_tp1 = getVelocity(cvs_k[idx].qe_tp1, cvs_k[idx].dfe, cvs_k[idx].slpe, cvs_k[idx].rc);
			//cvs_k[idx].vs_tp1 = getVelocity(cvs_k[idx].qs_tp1, cvs_k[idx].dfs, cvs_k[idx].slps, cvs_k[idx].rc);
			if (cvs_k[idx].dp_tp1 > dMinLimit) {
				setEffCells(cvs_k, idx);
			}
		}
		__syncthreads();  //sync_05
	}
}
//#endif

__host__ __device__ void calWFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx) {
	if (gvi_L.nCols == 1) { return; }
	flux flxw; //W, x-
	if (cvs_L[idx].colx == 0 || cvs_L[idx].cvidx_atW == -1)//w 측 경계셀
	{
		if (cvs_L[idx].isBCcell == 1) {
			flxw = noFlx(); // w측 최 경계에서는 w 방향으로 flx 없다.
		}
		else {// w측 최 경계에서는 w 방향으로 자유수면 flx 있다.
			double slp_tm1 = 0; // 2021.08.06. 인접셀 경사 적용하지 않고, bedslope 매개변수 값만 적용
			//if (cvs_L[idx].cvdix_atE >= 0)
			//{
			//	double he = cvs_L[cvs_L[idx].cvdix_atE].dp_t + cvsele_L[cvs_L[idx].cvdix_atE];
			//	double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
			//	slp_tm1 = (he - hcur) / gvi_L.dx; //i+1 셀과의 e 수면경사를 w 방향에 적용한다.
			//}
			slp_tm1 = slp_tm1 + gvi_L.domainOutBedSlope;
			if (slp_tm1 >= 0.0 && cvs_L[idx].dp_tp1 > dMinLimit) {
				// slp_tm1 > 0 인 경우가 아니면, w 방향으로 흐름 없다. e 방향으로 흐른다.
				flxw = calMEq_DWEm_Deterministric(cvs_L[idx].qw_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0.0);
			}
			else { flxw = noFlx(); }
		}
	}
	else {
		flxw.v = cvs_L[cvs_L[idx].cvidx_atW].ve_tp1; //2021.08.04
		flxw.wsslp = cvs_L[cvs_L[idx].cvidx_atW].slpe;
		flxw.q = cvs_L[cvs_L[idx].cvidx_atW].qe_tp1;
		flxw.dflow = cvs_L[cvs_L[idx].cvidx_atW].dfe;
	}
	cvs_L[idx].qw_tp1 = flxw.q;
	cvs_L[idx].vw_tp1 = flxw.v;
	//cvs_L[idx].slpw = flxw.wsslp;
	cvs_L[idx].dfw = flxw.dflow;
}

__host__ __device__ void calEFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx) {
	if (gvi_L.nCols == 1) { return; }
	flux flxe;    //E,  x+
	if (cvs_L[idx].colx == (gvi_L.nCols - 1) || cvs_L[idx].cvdix_atE == -1) {
		if (cvs_L[idx].isBCcell == 1) { flxe = noFlx(); }
		else {
			double slp_tm1 = 0.0;
			//if (cvs_L[idx].cvidx_atW >= 0) {
			//	double hw = cvs_L[cvs_L[idx].cvidx_atW].dp_t + cvsele_L[cvs_L[idx].cvidx_atW];
			//	double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
			//	slp_tm1 = (hcur - hw) / gvi_L.dx;
			//}
			slp_tm1 = slp_tm1 - gvi_L.domainOutBedSlope;
			if (slp_tm1 <= 0.0 && cvs_L[idx].dp_tp1 > dMinLimit) {
				// slp_tm1 < 0 인 경우가 아니면, e 방향으로 흐름 없다. w 방향으로 흐른다.
				flxe = calMEq_DWEm_Deterministric(cvs_L[idx].qe_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0.0);
			}
			else { flxe = noFlx(); }
		}
	}
	else {
		flxe = getFluxToEorS(cvs_L, cvsele_L, gvi_L, idx, cvs_L[idx].cvdix_atE, 1);
	}
	cvs_L[idx].ve_tp1 = flxe.v; //2021.08.04
	cvs_L[idx].dfe = flxe.dflow;
	cvs_L[idx].slpe = flxe.wsslp;
	cvs_L[idx].qe_tp1 = flxe.q;
}

__host__ __device__ void calNFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx) {
	if (gvi_L.nRows == 1) { return; }
	flux flxn;  //N, y-
	if (cvs_L[idx].rowy == 0 || cvs_L[idx].cvidx_atN == -1) {
		if (cvs_L[idx].isBCcell == 1) { flxn = noFlx(); }
		else {// n측 최 경계에서는 n 방향으로 자유수면 flx 있다.
			double slp_tm1 = 0.0;
			//if (cvs_L[idx].cvidx_atS >= 0) {
			//	//double slp_tm1 = (cvs[cvs[idx].cvaryNum_atS].hp_t - cvs[idx].hp_t) / gv.dx; //j+1 셀과의 수면경사를 w 방향에 적용한다.
			//	double hs = cvs_L[cvs_L[idx].cvidx_atS].dp_t + cvsele_L[cvs_L[idx].cvidx_atS];
			//	double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
			//	slp_tm1 = (hs - hcur) / gvi_L.dx;
			//}
			slp_tm1 = slp_tm1 + gvi_L.domainOutBedSlope;
			if (slp_tm1 >= 0.0 && cvs_L[idx].dp_tp1 > dMinLimit) {
				// slp_tm1 > 0 인 경우가 아니면, n 방향으로 흐름 없다. s 방향으로 흐른다.
				flxn = calMEq_DWEm_Deterministric(cvs_L[idx].qn_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0.0);
			}
			else { flxn = noFlx(); }
		}
	}
	else {
		flxn.v = cvs_L[cvs_L[idx].cvidx_atN].vs_tp1; //2021.08.04
		flxn.wsslp = cvs_L[cvs_L[idx].cvidx_atN].slps;
		flxn.dflow = cvs_L[cvs_L[idx].cvidx_atN].dfs;
		flxn.q = cvs_L[cvs_L[idx].cvidx_atN].qs_tp1;
	}
	cvs_L[idx].qn_tp1 = flxn.q;
	cvs_L[idx].vn_tp1 = flxn.v;
	cvs_L[idx].dfn = flxn.dflow;
}

__host__ __device__ void calSFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx) {
	if (gvi_L.nRows == 1) { return; }
	flux flxs;//S, y+
	if (cvs_L[idx].rowy == (gvi_L.nRows - 1)
		|| cvs_L[idx].cvidx_atS == -1) {
		if (cvs_L[idx].isBCcell == 1) { flxs = noFlx(); }
		else {
			double slp_tm1 = 0.0;
			//if (cvs_L[idx].cvidx_atN >= 0) {
			//	double hn = cvs_L[cvs_L[idx].cvidx_atN].dp_t + cvsele_L[cvs_L[idx].cvidx_atN];
			//	double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
			//	slp_tm1 = (hcur - hn) / gvi_L.dx;
			//}
			slp_tm1 = slp_tm1 - gvi_L.domainOutBedSlope;
			if (slp_tm1 <= 0.0 && cvs_L[idx].dp_tp1 > dMinLimit) {
				// slp_tm1 < 0 인 경우가 아니면, s 방향으로 흐름 없다. n 방향으로 흐른다.
				flxs = calMEq_DWEm_Deterministric(cvs_L[idx].qs_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0.0);
			}
			else { flxs = noFlx(); }
		}
	}
	else {
		flxs = getFluxToEorS(cvs_L, cvsele_L, gvi_L, idx, cvs_L[idx].cvidx_atS, 3);
	}
	cvs_L[idx].vs_tp1 = flxs.v; //2021.08.04
	cvs_L[idx].dfs = flxs.dflow;
	cvs_L[idx].slps = flxs.wsslp;
	cvs_L[idx].qs_tp1 = flxs.q;
}

__host__ __device__ flux calMEq_DWEm_Deterministric(double qt, 
	double dt_sec, double slp, double rc, double dflow, double qt_ip1){
	flux flx;
	double qapp = qt;
	//double q = (qapp - (gravity * dflow * dt_sec * slp)) /
	//         (1 + gravity * dt_sec * (rc * rc) * DeviceFunction.Sqrt((qapp * qapp + qt_ip1 * qt_ip1) / 2.0) 
	//         / pow(dflow, 7 / 3.0));

	double q = (qapp - (GRAVITY * dflow * dt_sec * slp)) /
		(1 + GRAVITY * dt_sec * (rc * rc) * abs(qapp) / pow(dflow, 7.0 / 3.0));

	flx.q = q;
	flx.v = flx.q / dflow;  // Manning 결과와 같다. flx.v = pow(dflow, 2 / 3.0) * sqrt(abs(slp)) / mN; 
	flx.dflow = dflow;
	flx.wsslp = slp;
	return flx; ;
}

//targetCellDir : E = 1, S = 3, W = 5, N = 7, NONE = 0
__host__ __device__ flux getFluxToEorS(cvatt* cvs_L, double* cvsele_L,
	globalVinner gvi_L, int idxc, int idxt, int targetCellDir)
{
	cvatt curCell = cvs_L[idxc];
	cvatt tarCell = cvs_L[idxt];
	double slp = 0.0;
	double dhtp1 = tarCell.hp_tp1 - curCell.hp_tp1;
	if (dhtp1 == 0.0) { return noFlx(); }
	if (dhtp1 > 0.0
		&& tarCell.dp_tp1 <=dMinLimit) {
		return noFlx();
	}
	if (dhtp1 < 0.0
		&& curCell.dp_tp1 <= dMinLimit) {
		return noFlx();
	}
	slp = dhtp1 / gvi_L.dx;
	if (abs(slp) <= slpMIN) { return noFlx(); }
	double dflow = fmax(curCell.hp_tp1, tarCell.hp_tp1)
		- fmax(cvsele_L[idxc], cvsele_L[idxt]);
	// 최대 수심법
	//dflow = DeviceFunction.Max(curCell.hp_tp1, tarCell.hp_tp1); 
	//// 수심평균 법
	//double maxBedElev = DeviceFunction.Max(curCell.elez, tarCell.elez);
	//double d1 = curCell.hp_tp1 - maxBedElev;
	//if (d1 < 0) { d1 = 0; }
	//double d2 = tarCell.hp_tp1 - maxBedElev;
	//if (d2 < 0) { d2 = 0; }
	//dflow = (d1 + d2) / 2.0;
	//// 수심평균 법
	if (dflow <= 0.0) { return noFlx(); }
	double qt = 0.0;
	double qtp1 = 0.0;
	double q_ip1 = 0.0;
	double u_ip1 = 0.0;
	if (targetCellDir == 1) {
		qt = curCell.qe_t;
		qtp1 = curCell.qe_tp1; 
		u_ip1 = tarCell.ve_tp1; q_ip1 = tarCell.qe_tp1;
	}
	else { // 1 or 3 이 들어온다.
		qt = curCell.qs_t;
		qtp1 = curCell.qs_tp1;
		u_ip1 = tarCell.vs_tp1; q_ip1 = tarCell.qs_tp1;
	}
	flux flx;
#ifdef isDWE
		flx = calMEq_DWE_Deterministric(qt, dflow,
			slp, curCell.rc, gvi_L.dx, gvi_L.dt_sec, q_ip1, u_ip1);
#else
		flx = calMEq_DWEm_Deterministric(qt,
			gvi_L.dt_sec, slp, curCell.rc, dflow, q_ip1);
#endif

#ifdef isAS
#else
		if (abs(flx.q) > 0.0) {
			flx = getFluxUsingSubCriticalCon(flx, gvi_L.froudeNCriteria);
			flx = getFluxUsingFluxLimit(flx, gvi_L.dx, gvi_L.dt_sec);
			//flx = getFluxqUsingFourDirLimitUsingDepthCondition(currentCell, flx, dflow, dx, dt_sec); //이건 수렴이 잘 안된다.
			//flx = getFluxUsingFourDirLimitUsingCellDepth(currentCell, targetCell, flx, dx, dt_sec);
			//flx = getFluxUsingFourDirLimitUsingDh(flx, dhtp1, dx, dt_sec); // 이건 소스에서 수심이 급격히 올라간다.
		}
#endif
	//flx.slp = slp;
	return flx;
}

__host__ __device__ flux calMEq_DWE_Deterministric(double qt, double dflow,
	double slp, double rc, float dx, double dt_sec, double q_ip1, double u_ip1)
{
	flux flx;
	double qapp = qt; //Math.Abs(qt);
	//2019.1.2 관성이 없을 경우에는 
	// slp가 + 면 q는 -, slp가 - 이면 q는 + 가 되어야 함.
	// 이전 t에서 q 가  0 이면, slp가 + 일때 무조건 q는 - , slp가 - 일때는 q는 무조건 +.
	// 이전 t에서 q 가  - 이면, slp가 + 일때 무조건 q는 - , slp가 - 일때는 q는 - 일수도 있고, + 일수도 있음. => 조건 처리 필요
	// 이전 t에서 q 가 + 이면, slp가 + 일때 q는 - 일수도 있고, + 일수도 있음, slp가 - 일때는 q는 무조건 +. => 조건 처리 필요

	double ut = qapp / dflow;
	
	double q = (qapp - (GRAVITY * dflow * dt_sec * slp)) /
		(1 + ut * dt_sec / dx + GRAVITY * dt_sec * (rc * rc) * abs(qapp) / pow(dflow, 7.0 / 3.0));
	//double q = ((qapp - q_ip1 * u_ip1 * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//                (1 - ut * dt_sec / dx + gravity * dt_sec * (rc * rc) * abs(qapp) 
	//                / DeviceFunction.Pow(dflow, 7.0 / 3.0)));
	//double q = ((qapp - Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2.0) * (u_ip1+ut)/2.0 * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//              (1 - (u_ip1 + ut) / 2.0 * dt_sec / dx + gravity * dt_sec * (rc * rc) * Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2.0) 
	//             / DeviceFunction.Pow(dflow, 7.0 / 3.0)));
	//double q = ((qapp - Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2.0) * (u_ip1 + ut) / 2.0 * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//               (1 - ut * dt_sec / dx + gravity * dt_sec * (rc * rc) * Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2.0) 
	//              / DeviceFunction.Pow(dflow, 7.0 / 3.0)));
	//double q = ((qapp - Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2.0) * ut * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//   (1 - ut * dt_sec / dx + gravity * dt_sec * (rc * rc) * qapp / DeviceFunction.Pow(dflow, 7.0 / 3.0)));

	flx.q = q;
	flx.v = flx.q / dflow;  // Manning 결과와 같다. flx.v = pow(dflow, 2 / 3) * sqrt(abs(slp)) / mN; 
	flx.dflow = dflow;
	flx.wsslp = slp;
	return flx; ;
}

__host__ __device__ flux getFluxUsingSubCriticalCon(flux inflx, float froudNCriteria){
	double v_wave = sqrt(GRAVITY * inflx.dflow);
	double fn = abs(inflx.v) / v_wave;
	if (fn > froudNCriteria) {
		double v = froudNCriteria * v_wave;
		if (inflx.q < 0.0) { v = -1.0 * v; }
		inflx.q = inflx.v * inflx.dflow;
		inflx.v = v;
	}
	return inflx;
}

__host__ __device__ flux getFluxUsingFluxLimit(flux inflx, 
	float dx, double dt_sec){
	double qmax = inflx.dflow * dx / 2.0 / dt_sec; // 수위차의 1/2 이 아니라, 흐름 수심의 1/2이므로, 수위 역전 될 수 있다.
	if (abs(inflx.q) > qmax) {
		if (inflx.q < 0.0) {
			inflx.q = -1.0 * qmax; 
		}
		else {
			inflx.q = qmax;
		}
		inflx.v = inflx.q / inflx.dflow;
	}
	return inflx;
}

__host__ __device__ void initializeThisStepAcell(cvatt* cvs_L, cvattAddAtt* cvsAA_L,
	bcAppinfo* bcAppinfos_L, double elev, double rfi_read_mPs_L,
	int idx, thisProcessInner psi_L, globalVinner gvi_L)
{
	double h = cvs_L[idx].dp_tp1 + elev;// cvs_L[idx].elez; //elev 가 변경되는 경우가 있으므로, 이렇게 수위설정
	if (cvs_L[idx].hp_tp1 <= h) { // 지면고가 높아진 경우
		// dem  고도 변경되면, 수심이 바뀐다. 수위는 유지.
		// cvs_L[idx].hp_t=cvs_L[idx].elez + cvs_L[idx].dp_t 이므로, cvs_L[idx].dp_t 이값과 cvs_L[idx].dp_tp1  모두 업데이트 해줘야 한다.
		cvs_L[idx].dp_tp1 = cvs_L[idx].hp_tp1 - elev;
		if (cvs_L[idx].dp_tp1 < 0.0) { cvs_L[idx].dp_tp1 = 0.0; }
		cvs_L[idx].dp_t = cvs_L[idx].dp_tp1;
	}
	else {
		cvs_L[idx].dp_t = cvs_L[idx].dp_tp1;
	}
	cvs_L[idx].qe_t = cvs_L[idx].qe_tp1;
	cvs_L[idx].qw_t = cvs_L[idx].qw_tp1;
	cvs_L[idx].qs_t = cvs_L[idx].qs_tp1;
	cvs_L[idx].qn_t = cvs_L[idx].qn_tp1;
	double sourceAlltoRoute_tp1_dt_m = 0.0;
	if (cvs_L[idx].isBCcell == 1) { // prj.isbcApplied == 1 조건은 보장됨
		int bci = getBcAppinfoidx(bcAppinfos_L, gvi_L.bcCellCountAll, idx);
		bcAppinfos_L[bci].bcDepth_dt_m_tp1 = getCDasDepthWithLinear(bcAppinfos_L[bci].bctype,
			bcAppinfos_L[bci].bcData_curOrder, bcAppinfos_L[bci].bcData_nextOrder,
			bcAppinfos_L[bci].bcData_curOrderStartedTime_sec, elev, psi_L.tnow_sec, gvi_L);
		if (bcAppinfos_L[bci].bctype == 1)//1:  Discharge,  2: Depth, 3: WaterLevel,  4: None
		{//경계조건이 유량일 경우, 소스항에 넣어서 홍수추적한다. 수심으로 환산된 유량..
			sourceAlltoRoute_tp1_dt_m = bcAppinfos_L[bci].bcDepth_dt_m_tp1;
		}
		else
		{//경계조건이 유량이 아닐경우, 홍수추적 하지 않고, 고정된 값 적용.
			cvs_L[idx].dp_tp1 = bcAppinfos_L[bci].bcDepth_dt_m_tp1;
			if (psi_L.tnow_sec == 0.0) {
				cvs_L[idx].dp_t = cvs_L[idx].dp_tp1;
			}
		}
	}
	cvsAA_L[idx].sourceRFapp_dt_meter = 0.0;
	//-1, 0 :false, 1: true
	if (psi_L.isRFApplied == 1 && psi_L.rfEnded == 0.0)
	{
		if (psi_L.rfType == weatherDataType::Raster_ASC) {
			cvsAA_L[idx].sourceRFapp_dt_meter = rfi_read_mPs_L * gvi_L.dt_sec;
		}
		else {
			cvsAA_L[idx].sourceRFapp_dt_meter = psi_L.rfReadintensityForMAP_mPs * gvi_L.dt_sec;
		}
	}
	sourceAlltoRoute_tp1_dt_m = sourceAlltoRoute_tp1_dt_m + cvsAA_L[idx].sourceRFapp_dt_meter;
	cvs_L[idx].dp_t = cvs_L[idx].dp_t + sourceAlltoRoute_tp1_dt_m;
	cvs_L[idx].dp_tp1 = cvs_L[idx].dp_tp1 + sourceAlltoRoute_tp1_dt_m;
	cvs_L[idx].hp_tp1 = cvs_L[idx].dp_tp1 + elev;
}

__host__ __device__ void setStartingConditionCVs_inner(cvatt* cvs_L, cvattAddAtt* cvsAA_L,
	double* cvselez_L, int idx) {
	cvs_L[idx].dp_t = cvsAA_L[idx].initialConditionDepth_m;
	cvs_L[idx].dp_tp1 = cvs_L[idx].dp_t;
	cvs_L[idx].ve_tp1 = 0.0;
	cvs_L[idx].qe_tp1 = 0.0;
	cvs_L[idx].qw_tp1 = 0.0;
	cvs_L[idx].qn_tp1 = 0.0;
	cvs_L[idx].qs_tp1 = 0.0;
	//cvs_L[idx].hp_tp1 = cvs_L[idx].dp_tp1 + cvs_L[idx].elez;
	cvs_L[idx].hp_tp1 = cvs_L[idx].dp_tp1 + cvselez_L[idx];
	//cvsAA_L[idx].fdmaxV = 0;//E = 1, S = 3, W = 5, N = 7, NONE = 0
	//cvsAA_L[idx].bcData_curOrder = 0;
	cvsAA_L[idx].sourceRFapp_dt_meter = 0.0;
	cvsAA_L[idx].rfAccCell = 0.0;
	cvsAA_L[idx].saturatedByCellRF = 0.0;
	//cvsAA_L[idx].rfReadintensity_mPsec = 0;
	cvs_L[idx].isSimulatingCell = 0;
}

__host__ __device__ double getCDasDepthWithLinear(int bctype, double vcurOrder, double vnextOrder,
	int t_curOrderStarted_sec, double elev_m, double tnow_sec, globalVinner gvi_L)
{
	double valueAsDepth_curOrder = 0.0;
	double valueAsDepth_nextOrder = 0.0;
	double dx = gvi_L.dx;
	double dt_s = gvi_L.dt_sec;
	//1:  Discharge,  2: Depth, 3: WaterLevel,  4: None
	switch (bctype)
	{
	case 1://conditionDataType::Discharge:
		valueAsDepth_curOrder = (vcurOrder / dx / dx) * dt_s;
		valueAsDepth_nextOrder = (vnextOrder / dx / dx) * dt_s;
		break;
	case 2://conditionDataType::Depth:
		valueAsDepth_curOrder = vcurOrder;
		valueAsDepth_nextOrder = vnextOrder;
		break;
	case 3://conditionDataType::WaterLevel:
		valueAsDepth_curOrder = vcurOrder - elev_m;
		valueAsDepth_nextOrder = vnextOrder - elev_m;
		break;
	}
	if (valueAsDepth_curOrder < 0.0) { valueAsDepth_curOrder = 0.0; }
	if (valueAsDepth_nextOrder < 0.0) { valueAsDepth_nextOrder = 0.0; }
	double bcDepth_dt_m_tp1 = 0.0;
#ifndef isAS // 해석해 테스트가 아닐때는 이 조건 사용
		bcDepth_dt_m_tp1 = (valueAsDepth_nextOrder - valueAsDepth_curOrder)
			* (tnow_sec - t_curOrderStarted_sec) / gvi_L.dtbc_sec
			+ valueAsDepth_curOrder;
#else
		bcDepth_dt_m_tp1 = valueAsDepth_curOrder; // 해석해 테스트는 이 조건
#endif
	return  bcDepth_dt_m_tp1;
}

__host__ __device__ int getBcAppinfoidx(bcAppinfo * bcAppinfos, int bcCellCountAll, int cvidxToGet) {

	for (int i = 0; i < bcCellCountAll; ++i) {
		if (bcAppinfos[i].cvidx == cvidxToGet) {
			return i;
		}
	}
	return -1;
}

__host__ __device__ void setEffCells(cvatt * cvs_L, int idx)
{
	cvs_L[idx].isSimulatingCell = 1;
	if (cvs_L[idx].cvdix_atE >= 0) {
		cvs_L[cvs_L[idx].cvdix_atE].isSimulatingCell = 1;
	}
	if (cvs_L[idx].cvidx_atW >= 0) {
		cvs_L[cvs_L[idx].cvidx_atW].isSimulatingCell = 1;
	}
	if (cvs_L[idx].cvidx_atN >= 0) {
		cvs_L[cvs_L[idx].cvidx_atN].isSimulatingCell = 1;
	}
	if (cvs_L[idx].cvidx_atS >= 0) {
		cvs_L[cvs_L[idx].cvidx_atS].isSimulatingCell = 1;
	}
}

// E에서 시작해서 시계방향으로 첫번째 최대값 방향 반환
__host__ __device__ fluxNfd get_maxFlux_FD(cvatt* cvs_L, int i) {
	fluxNfd flxmxfd;
	double ve = cvs_L[i].ve_tp1;
	double vs = cvs_L[i].vs_tp1;
	double avw = abs(cvs_L[i].vw_tp1);
	double avn = abs(cvs_L[i].vn_tp1);
	double vmax = 0.0;
	int fdMaxV = 0; //	flowDirection8G2D :: E1 = 1, SE2 = 2, S3 = 3, SW4 = 4, W5 = 5, NW6 = 6, N7 = 7, NE8 = 8, NONE = 0
	if (ve > 0) {
		vmax = ve;
		fdMaxV = 1;
	}
	if (cvs_L[i].vw_tp1 < 0 && avw > vmax) { // 이경우가 셀 밖으로 유속 있는 경우
		vmax = avw;
		fdMaxV = 5;
	}
	if (vs > vmax) { // 이러면 vs>0 보장
		vmax = vs;
		fdMaxV = 3;
	}
	if (cvs_L[i].vn_tp1 < 0 && avn > vmax) {// 이경우가 셀 밖으로 유속 있는 경우
		vmax = avn;
		fdMaxV = 7;
	}
	flxmxfd.fd_maxv = fdMaxV;
	flxmxfd.v = vmax;

	// 여기서 부터 최대 흐름 수심 찾고, 최대 유량 방향
	if (vmax == 0) {
		flxmxfd.dflow = 0;
		flxmxfd.q = 0;
		flxmxfd.fd_maxq = 0; //flowDirection8G2D::NONE ==0
	}
	else {

		double dflow_max = 0.0;
		dflow_max = cvs_L[i].dfe;
		if (cvs_L[i].dfw > dflow_max) {
			dflow_max = cvs_L[i].dfw;
		}
		if (cvs_L[i].dfn > dflow_max) {
			dflow_max = cvs_L[i].dfn;
		}
		if (cvs_L[i].dfs > dflow_max) {
			dflow_max = cvs_L[i].dfs;
		}
		flxmxfd.dflow = dflow_max; // 여기. 최대 흐름 수심

		double qe = cvs_L[i].qe_tp1;
		double qs = cvs_L[i].qs_tp1;
		double aqw = abs(cvs_L[i].qw_tp1);
		double aqn = abs(cvs_L[i].qn_tp1);
		double qmax = 0.0;
		int fd_maxq = 0; //	flowDirection8G2D :: E1 = 1, SE2 = 2, S3 = 3, SW4 = 4, W5 = 5, NW6 = 6, N7 = 7, NE8 = 8, NONE = 0

		if (qe > 0) {
			qmax = qe;
			fd_maxq = 1;
		}
		if (cvs_L[i].qw_tp1 < 0 && aqw>qmax) { // 이경우가 셀 밖으로 유량 있는 경우
			qmax = aqw;
			fd_maxq = 5;
		}
		if (qs > qmax) { // 이러면 vs>0 보장
			qmax = qs;
			fd_maxq = 3;
		}
		if (cvs_L[i].qn_tp1 < 0 && aqn>qmax) {// 이경우가 셀 밖으로 유량 있는 경우
			qmax = aqn;
			fd_maxq = 7;
		}
		flxmxfd.q = qmax;
		flxmxfd.fd_maxq = fd_maxq; //flowDirection8G2D::NONE ==0
	}
	return flxmxfd;
}


__host__ __device__ double getVNConditionValue(cvatt* cvs_L, int i) {
	double searchMIN = DBL_MAX;
	double curValue = 0.0;
	double rc = cvs_L[i].rc;
	// e 값과 중복되므로, w는 계산하지 않는다.
	if (cvs_L[i].dfe > 0.0) {
		searchMIN = 2.0 * rc * sqrt(abs(cvs_L[i].slpe))
			/ pow(cvs_L[i].dfe, 5.0 / 3.0);
	}
	// s 값과 중복되므로, n는 계산하지 않는다.
	if (cvs_L[i].dfs > 0.0) {
		curValue = 2.0 * rc * sqrt(abs(cvs_L[i].slps))
			/ pow(cvs_L[i].dfs, 5.0 / 3.0);
		if (curValue < searchMIN) {
			searchMIN = curValue;
		}
	}
	return searchMIN;
}

// 2021.08.05 NR 밖에서 이것을 이용하면, 각 함수에서 v 계산하는 것 보다 느려진다.
__host__ __device__ double getVelocity(double q, double dflow, double slp, double rc) {
	if (dflow <= 0.0) {
		return 0;
	}
	else {
		double v1 = q / dflow;  // dflow가 아주 작은 경우 발산한다.
		return v1;

	}
	//double v2 = pow(dflow, 2 / 3.0) * sqrt(abs(slp)) / rc;	 // 항상 +
	 //double v2 = sqrt(GRAVITY * dflow); // 항상 +
	 //double v = min(abs(v1), v2); // 항상 +
	 ////if (v > 9) {
	 ////	int a = 1;
	 ////}
	 //if (q < 0) {
	 //	v = -1.0 * v;
	 //}
	 //return v;
}