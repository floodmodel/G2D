#include "hip/hip_runtime.h"
﻿#ifdef __INTELLISENSE___
void __syncthreads();
#else
void __syncthreads();
#endif

#include "stdafx.h"
#include <ATLComTime.h>

#include "g2d.h"
#include "g2d_cuda.cuh"

using namespace std;
namespace fs = std::filesystem;

extern fs::path fpn_log;

extern projectFile prj;
extern domaininfo di;
extern domainCell** dmcells;
extern generalEnv ge;

extern cvatt* cvs;
extern cvattAddAtt* cvsAA;
extern double* cvsele;
extern double* rfi_read_mPs;
extern bcAppinfo * bcAppinfos;

extern thisProcess ps;
extern thisProcessInner psi;
extern globalVinner gvi;
extern dataForCalDT dataForDT;
extern minMaxCVidx mnMxCVidx;


#ifdef OnGPU
int simulationControl_GPU()
{
	hipDeviceProp_t m_deviceProp;
	hipGetDeviceProperties(&m_deviceProp, 0);
	if (!m_deviceProp.canMapHostMemory) {
		fprintf(stderr, "Device does not support mapping CPU host memory!\n");
		return 0;
	}
	double simDur_min = prj.simDuration_min + 1.0;
	int bcDataOrder = 0;
	int rfDataOrder = 0;
	int demToChangeEnded = 1;
	double tnow_min_bak = 0;
	int dtbc_min = prj.bcDataInterval_min;
	if (prj.isDEMtoChangeApplied == 1) {
		demToChangeEnded = 0;
	}
	initThisProcess();
	initGlobalVinner();
	initFloodingThresholds();
	initMinMax();
	cvatt* d_cvs;
	cvattAddAtt* d_cvsAA;
	double* d_cvsele;
	double* d_rfi_read_mPs;
	double* d_dtsec;
	bcAppinfo* d_bcAppinfos;
	minMaxCVidx* d_minMaxCVidx;

	size_t  ms_cvs_ncvs = di.cellNnotNull * sizeof(cvatt);
	size_t  ms_cvsAA_ncvs = di.cellNnotNull * sizeof(cvattAddAtt);
	size_t ms_double_ncvs = di.cellNnotNull * sizeof(double);
	size_t ms_bcAppinfo_allBCcells = prj.bcCellCountAll * sizeof(bcAppinfo);
	size_t ms_minMaxCVidx = sizeof(minMaxCVidx);
	size_t ms_minMaxCVidx_TPB = ps.threadsPerBlock*sizeof(minMaxCVidx);

	dim3 thPblock(ps.threadsPerBlock, 1, 1); //blockDim    
	dim3 bPgrid(di.cellNnotNull / (thPblock.x * thPblock.y) + 1, 1); //gridDim 

	//clock_t  ts, tf;
	//long tc_memcopy = 0;
	//long tc_setStartingConditionCVs_GPU = 0;
	//long tc_set_rf_bc_dem = 0;
	//long tc_initilizeThisStep_GPU = 0;
	//long tc_GSiteration = 0;
	//long tc_getMinMaxFromCV = 0;
	//long tc_getMinMax_reduction = 0;
	//long tc_memcpy_mnMxCVidx = 0;
	//long tc_setAllCVFalse = 0;

	//ts = clock();
	hipMalloc((void**)& d_cvs, ms_cvs_ncvs);
	hipMalloc((void**)& d_cvsAA, ms_cvsAA_ncvs);
	hipMalloc((void**)& d_cvsele, ms_double_ncvs);
	hipMalloc((void**)& d_bcAppinfos, ms_bcAppinfo_allBCcells);
	hipMalloc((void**)& d_rfi_read_mPs, ms_double_ncvs);
	hipMalloc((void**)& d_minMaxCVidx, bPgrid.x * sizeof(minMaxCVidx));
	hipMalloc((void**)& d_dtsec, sizeof(double));
	hipMemcpy(d_cvs, cvs, ms_cvs_ncvs, hipMemcpyHostToDevice);
	hipMemcpy(d_cvsAA, cvsAA, ms_cvsAA_ncvs, hipMemcpyHostToDevice);
	hipMemcpy(d_cvsele, cvsele, ms_double_ncvs, hipMemcpyHostToDevice);
	hipMemcpy(d_bcAppinfos, bcAppinfos, ms_bcAppinfo_allBCcells, hipMemcpyHostToDevice);
	hipMemcpy(d_rfi_read_mPs, rfi_read_mPs, ms_double_ncvs, hipMemcpyHostToDevice);
	//tf = clock();
	//tc_memcopy = long(tf - ts);

	//ts = clock();
	setStartingConditionCVs_GPU << < bPgrid, thPblock >> > (d_cvs, d_cvsAA,
		d_cvsele, gvi.nCellsInnerDomain);
	//tf = clock();
	//tc_setStartingConditionCVs_GPU = long(tf - ts);

	do { //모의 시작할 때 t 는 초기 조건, t+dt는 소스 하나가 적용된 결과
		psi.tnow_min = psi.tnow_sec / 60.0;
		//ts = clock();
		if (prj.isbcApplied == 1) {//경계조건 등
			int bc_min = bcDataOrder * dtbc_min;
			if (((tnow_min_bak < bc_min) & (psi.tnow_min >= bc_min))
				|| bc_min == 0) {
				bcDataOrder++;
				getCellCD(bcDataOrder, dtbc_min); // 경계조건 값은 포인터로 넘긴다..
				hipMemcpy(d_bcAppinfos, bcAppinfos, ms_bcAppinfo_allBCcells, hipMemcpyHostToDevice);
			}
		}
		if (prj.isRainfallApplied == 1 && psi.rfEnded == 0) {//강우
			int rf_min = rfDataOrder * prj.rainfallDataInterval_min;
			if (((tnow_min_bak < rf_min) & (psi.tnow_min >= rf_min))
				|| rf_min == 0) {
				rfDataOrder++; //1부터 시작. 배열은 rainfallDataOrder-1
				psi.rfEnded = readRainfallAndGetIntensity(rfDataOrder);
				hipMemcpy(d_rfi_read_mPs, rfi_read_mPs, ms_double_ncvs, hipMemcpyHostToDevice);
			}
		}
		if (prj.isDEMtoChangeApplied == 1 && demToChangeEnded == 0) {//dem file 교체
			demToChangeEnded = changeDomainElevWithDEMFile(psi.tnow_min, tnow_min_bak);
			hipMemcpy(d_cvsele, cvsele, ms_double_ncvs, hipMemcpyHostToDevice);
		}
		//tf = clock();
		//tc_set_rf_bc_dem = long(tf - ts);

		//ts = clock();
		initilizeThisStep_GPU << < bPgrid, thPblock >> > (d_cvs, d_cvsAA,
			d_cvsele, d_bcAppinfos, d_rfi_read_mPs, psi, gvi);
		hipDeviceSynchronize();
		//tf = clock();
		//tc_initilizeThisStep_GPU = long(tf - ts);

		//ts = clock();
		runSolver_GPU << < bPgrid, thPblock >> > (d_cvs, d_bcAppinfos, d_cvsele, gvi);
		hipDeviceSynchronize();
		//tf = clock();
		//tc_GSiteration = long(tf - ts);
		
		//ts = clock();
		getMinMaxFromCV << < bPgrid, thPblock , ms_minMaxCVidx_TPB >> > (d_cvs,
			d_cvsAA, gvi, d_minMaxCVidx);
		hipDeviceSynchronize();
		//tf = clock();
		//tc_getMinMaxFromCV = long(tf - ts);

		// reduction=================================
		//ts = clock();
		if (bPgrid.x > 1) {
			int array_size = bPgrid.x;
			int numBlock = (array_size + thPblock.x - 1) / thPblock.x + 1;;
			while (numBlock != 1) {
				if (array_size < ps.threadsPerBlock) {
					numBlock = 1;
					//getMinMaxFromArray << < numBlock, thPblock >> > (d_minMaxCVidx, array_size, gvi, d_minMaxCVidx);
					getMinMaxFromArray << < numBlock, thPblock, ms_minMaxCVidx_TPB >> > (d_minMaxCVidx,
						array_size, gvi, d_minMaxCVidx);
					hipDeviceSynchronize(); // 커널함수 들이 완료될때 까지 대기, block 동기화
					break;
				}
				else {
					getMinMaxFromArray << < numBlock, thPblock , ms_minMaxCVidx_TPB >> > (d_minMaxCVidx,
						array_size, gvi, d_minMaxCVidx);
					hipDeviceSynchronize();// 커널함수 들이 완료될때 까지 대기, block 동기화 
					array_size = numBlock;
					numBlock = (numBlock + thPblock.x - 1) / thPblock.x + 1;

				}
			}
		}
		//tf = clock();
		//tc_getMinMax_reduction = long(tf - ts);

		//==========================================
		if (psi.tnow_sec >= psi.tsec_targetToprint) {
			hipMemcpy(cvs, d_cvs, ms_cvs_ncvs, hipMemcpyDeviceToHost);
			hipMemcpy(cvsAA, d_cvsAA, ms_cvsAA_ncvs, hipMemcpyDeviceToHost);
			//ts = clock();
			//if (prj.isFixedDT == 1) {
			//	hipMemcpy(&mnMxCVidx, d_minMaxCVidx, ms_minMaxCVidx, hipMemcpyDeviceToHost); // FTS인 경우, dt를 계산하려면, 매번 받아와야 한다. 
			//}
			//tf = clock();
			//tc_memcpy_mnMxCVidx = long(tf - ts);
			updateSummaryAndSetAllFalse();// 출력할때 마다 이 정보 업데이트

			//ts = clock();
			setAllCVFalse << <bPgrid, thPblock >> > (d_cvs, gvi.nCellsInnerDomain); // cvs.isSimulatingCell 을 복사하지 않고, 여기서 d_cvs에서 설정해 준다.
			hipDeviceSynchronize();
			//tf = clock();
			//tc_setAllCVFalse = long(tf - ts);
			//cout << "\n\nCurrent min           : " << psi.tnow_min << "min" << endl;
			//cout << "Time consumed malloc, memcopy          : " << tc_memcopy << "ms" << endl;
			//cout << "Time consumed setStartingCondition_GPU : " << tc_setStartingConditionCVs_GPU << "ms" << endl;
			//cout << "Time consumed tc_set_rf_bc_dem         : " << tc_set_rf_bc_dem << "ms" << endl;
			//cout << "Time consumed tc_initilizeThisStep_GPU : " << tc_initilizeThisStep_GPU << "ms" << endl;
			//cout << "Time consumed tc_GSiteration           : " << tc_GSiteration << "ms" << endl;
			//cout << "Time consumed tc_getMinMaxFromCV       : " << tc_getMinMaxFromCV << "ms" << endl;
			//cout << "Time consumed tc_getMinMax_reduction   : " << tc_getMinMax_reduction << "ms" << endl;
			//cout << "Time consumed tc_memcpy_mnMxCVidx      : " << tc_memcpy_mnMxCVidx << "ms" << endl;
			//cout << "Time consumed tc_setAllCVFalse         : " << tc_setAllCVFalse << "ms" << endl;
			makeOutputFiles(psi.tnow_sec, gvi.iGSmaxLimit);
			int progressRatio = (int)(psi.tnow_min / prj.simDuration_min * 100);
			printf("\rCurrent progress[min]: %d/%d[%d%%]..", (int)psi.tnow_min,
				(int)prj.simDuration_min, progressRatio);

			//한번 출력할때 마다 모의변수 업데이트
			if (updateProjectParameters() == 0) {
				return 0;
			}
			else if (prj.parChanged == 1) {
				initThisProcess();
				initGlobalVinner();
				initFloodingThresholds();
				simDur_min = prj.simDuration_min + 1.0;
			}
			psi.tsec_targetToprint = psi.tsec_targetToprint + ps.dt_printout_sec;
			ps.thisPrintStepStartTime = COleDateTime::GetCurrentTime();
		}
		tnow_min_bak = psi.tnow_min;
		psi.tnow_sec = psi.tnow_sec + gvi.dt_sec;
		if (prj.isFixedDT == 0) {
			//ts = clock();
			hipMemcpy(&mnMxCVidx, d_minMaxCVidx, ms_minMaxCVidx, hipMemcpyDeviceToHost); // dt를 계산하려면, 매번 받아와야 한다. 
			hipMemcpy(bcAppinfos, d_bcAppinfos, ms_bcAppinfo_allBCcells, hipMemcpyDeviceToHost); // dt를 계산하려면, 매번 받아와야 한다. 																						   
			//tf = clock();
			//tc_memcpy_mnMxCVidx = long(tf - ts);
  		   gvi.dt_sec = getDTsecWithConstraints(dataForDT, gvi, psi.tnow_sec, bcAppinfos,
				mnMxCVidx);
		}
	} while (psi.tnow_min < simDur_min);

	hipFree(d_cvs);
	hipFree(d_cvsAA);
	hipFree(d_cvsele);
	hipFree(d_bcAppinfos);
	hipFree(d_rfi_read_mPs);
	CUDA_CHECK(hipGetLastError());
	return 1;
}

__global__ void initilizeThisStep_GPU(cvatt* d_cvs, cvattAddAtt* d_cvsAA, double * d_cvsele,
	bcAppinfo* d_bcApp, double* d_rfi_read_mPs, 
	thisProcessInner psi_k, globalVinner gvi_k){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < gvi_k.nCellsInnerDomain) {
		initializeThisStepAcell(d_cvs, d_cvsAA, d_bcApp, d_cvsele[idx], d_rfi_read_mPs[idx], idx, psi_k, gvi_k);
		if (d_cvs[idx].dp_tp1 >dMinLimit) {/*gvi_k.dMinLimitforWet*/
			setEffCells(d_cvs, idx);
		}
	}
}

__global__ void setStartingConditionCVs_GPU(cvatt* d_cvs, cvattAddAtt * d_cvsAA, 
	double* d_cvsele, int arraySize) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < arraySize) {
		setStartingConditionCVs_inner(d_cvs, d_cvsAA, d_cvsele, tid);
	}
}

__global__ void getMinMaxFromCV(cvatt* cvs_k, cvattAddAtt* cvsAA_k,
	globalVinner gvi_k, minMaxCVidx* odata) {
	/*__shared__ minMaxCVidx sdata[THPB]; */
	extern __shared__ minMaxCVidx sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	sdata[tid].dflowmaxInThisStep = -9999; // 여기에서 초기화 해준다. 실제 배열 길이가 < tid + s 인경우에도 값을 입력..
	sdata[tid].vmaxInThisStep =-9999;
	sdata[tid].VNConMinInThisStep = 9999;
	__syncthreads();
	if (idx < gvi_k.nCellsInnerDomain) {
		fluxData flxmax;
		flxmax = getFD4MaxValues(cvs_k, idx);
		cvsAA_k[idx].fdmaxV = flxmax.fd;
		cvsAA_k[idx].vmax = flxmax.v;
		cvsAA_k[idx].Qmax_cms = flxmax.q * gvi_k.dx;
		__syncthreads(); // 초기화 부분에서 최대한 sync 시킨다. // 필수. 중요
		sdata[tid].dflowmaxInThisStep = flxmax.dflow;
		sdata[tid].vmaxInThisStep = flxmax.v;
		__syncthreads(); // 초기화 부분에서 최대한 sync 시킨다. // 필수. 중요
		if (gvi_k.isApplyVNC == 1) {
			sdata[tid].VNConMinInThisStep = getVNConditionValue(cvs_k, idx);
		}
		__syncthreads(); // 초기화 부분에서 최대한 sync 시킨다. // 필수. 중요
		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if (tid < s ) {
				if (sdata[tid].dflowmaxInThisStep < sdata[tid + s].dflowmaxInThisStep) {
					sdata[tid].dflowmaxInThisStep = sdata[tid + s].dflowmaxInThisStep;
					__syncthreads();
				}
				if (sdata[tid].vmaxInThisStep < sdata[tid + s].vmaxInThisStep) {
					sdata[tid].vmaxInThisStep = sdata[tid + s].vmaxInThisStep;
					__syncthreads();
				}
				if (gvi_k.isApplyVNC == 1) {
					if (sdata[tid].VNConMinInThisStep > sdata[tid + s].VNConMinInThisStep) {
						sdata[tid].VNConMinInThisStep = sdata[tid + s].VNConMinInThisStep;
						__syncthreads();
					}
				}
			}
			__syncthreads();
		}
	}
	if (tid == 0) {
		odata[blockIdx.x].dflowmaxInThisStep = sdata[0].dflowmaxInThisStep;
		odata[blockIdx.x].vmaxInThisStep = sdata[0].vmaxInThisStep;
		odata[blockIdx.x].VNConMinInThisStep = sdata[0].VNConMinInThisStep;
		__syncthreads();
	}
}
__global__ void getMinMaxFromArray(minMaxCVidx* minMaxCVidx_k, int arraySize,
	globalVinner gvi_k, minMaxCVidx* odata) {
	//__shared__ minMaxCVidx sdata[THPB];
	extern  __shared__ minMaxCVidx sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	sdata[tid].dflowmaxInThisStep = -9999; // 여기에서 초기화 해준다. 실제 배열 길이가 < tid + s 인경우에도 값을 입력..
	sdata[tid].vmaxInThisStep = -9999;
	sdata[tid].VNConMinInThisStep = 9999;
	__syncthreads();
	if (idx < arraySize) {
		sdata[tid].dflowmaxInThisStep = minMaxCVidx_k[idx].dflowmaxInThisStep;
		sdata[tid].vmaxInThisStep = minMaxCVidx_k[idx].vmaxInThisStep;
		sdata[tid].VNConMinInThisStep = minMaxCVidx_k[idx].VNConMinInThisStep;
		__syncthreads(); // 초기화 부분에서 최대한 sync 시킨다. // 필수. 중요

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if (tid < s) {
				if (sdata[tid].dflowmaxInThisStep < sdata[tid + s].dflowmaxInThisStep) {
					sdata[tid].dflowmaxInThisStep = sdata[tid + s].dflowmaxInThisStep;
					__syncthreads();
				}
				if (sdata[tid].vmaxInThisStep < sdata[tid + s].vmaxInThisStep) {
					sdata[tid].vmaxInThisStep = sdata[tid + s].vmaxInThisStep;
					__syncthreads();
				}
				if (gvi_k.isApplyVNC == 1) {
					if (sdata[tid].VNConMinInThisStep > sdata[tid + s].VNConMinInThisStep) {
						sdata[tid].VNConMinInThisStep = sdata[tid + s].VNConMinInThisStep;
						__syncthreads();
					}
				}
			}
			__syncthreads();
		}
	}
	if (tid == 0) {
		odata[blockIdx.x].dflowmaxInThisStep = sdata[0].dflowmaxInThisStep;
		odata[blockIdx.x].vmaxInThisStep = sdata[0].vmaxInThisStep;
		odata[blockIdx.x].VNConMinInThisStep = sdata[0].VNConMinInThisStep;
		__syncthreads();
	}
}

__global__ void setAllCVFalse(cvatt* d_cvs, int arraySize) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < arraySize) {
		d_cvs[tid].isSimulatingCell = 0;
		__syncthreads();
	}
}

//__global__ void calCEqUsingNR_Launcher(cvatt* cvs_k, bcAppinfo* bcAppinfos_k, double* cvsele_k,
//	globalVinner gvi_k) {
//	int nCells = gvi_k.nCellsInnerDomain;
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	if (idx < nCells) {
//		if (cvs_k[idx].isSimulatingCell == 1) {
//			calCEqUsingNR(cvs_k, gvi_k, bcAppinfos_k, cvsele_k, idx);
//			if (cvs_k[idx].dp_tp1 > dMinLimit) {
//				setEffCells(cvs_k, idx);
//			}
//			//__syncthreads();
//		}
//		__syncthreads();
//	}
//}

__global__ void runSolver_GPU(cvatt* cvs_k, bcAppinfo* bcAppinfos_k,
	double* cvsele_k, globalVinner gvi_k) {
	int nCells = gvi_k.nCellsInnerDomain;
	int igsLimit = gvi_k.iGSmaxLimit;
	for (int igs = 0; igs < igsLimit; ++igs) {
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		if (idx < nCells) {
			if (cvs_k[idx].isSimulatingCell == 1) {
				calCEqUsingNR_DEVICE(cvs_k, gvi_k, bcAppinfos_k, cvsele_k, idx);
				if (cvs_k[idx].dp_tp1 > dMinLimit) {
					setEffCells(cvs_k, idx);
				}
			}
			__syncthreads();
		}
	}
}

#endif

// 0 : 미수렴, 1: 수렴. __syncthreads(); 를 위해서 device 용을 하나더 만들었다. OnGPU 에서도 CPU 사용 가능하도록 함
__device__ int calCEqUsingNR_DEVICE(cvatt* cvs_L, globalVinner gvi_L,
	bcAppinfo* bcAppinfos_L, double* cvsele_L, int i) {
	double dp_old = cvs_L[i].dp_tp1;
	for (int inr = 0; inr < gvi_L.iNRmaxLimit; ++inr) {
		double c1_IM = gvi_L.dt_sec / gvi_L.dx;
		double dn = cvs_L[i].dp_tp1;
		__syncthreads();
		calWFlux(cvs_L, cvsele_L, gvi_L, i);
		__syncthreads();
		calEFlux(cvs_L, cvsele_L, gvi_L, i);
		__syncthreads();
		calNFlux(cvs_L, cvsele_L, gvi_L, i);
		__syncthreads();
		calSFlux(cvs_L, cvsele_L, gvi_L, i);
		__syncthreads();
		// 현재 셀의 수위가 올라가려면  -> qe-, qw+, qs-, qn+
		double dnp1 = 0.0;
		double fn = dn - cvs_L[i].dp_t + (cvs_L[i].qe_tp1 - cvs_L[i].qw_tp1
			+ cvs_L[i].qs_tp1 - cvs_L[i].qn_tp1) * c1_IM;//- sourceTerm; //이건 음해법
		double eElem = pow(cvs_L[i].dfe, 2 / 3.0) * sqrt(abs(cvs_L[i].slpe)) / cvs_L[i].rc;
		double sElem = pow(cvs_L[i].dfs, 2 / 3.0) * sqrt(abs(cvs_L[i].slps)) / cvs_L[i].rc;
		double dfn = 1 + (eElem + sElem) * (5.0 / 3.0) * c1_IM;// 이건 음해법
		if (dfn == 0) { break; }
		dnp1 = dn - fn / dfn;
		if (cvs_L[i].isBCcell == 1) {
			int bcidx = getBcAppinfoidx(bcAppinfos_L, gvi_L.bcCellCountAll, i);
			if (bcAppinfos_L[bcidx].bctype == 2) {// 1:Discharge, 2:Depth, 3:Height, 4:None
				dnp1 = bcAppinfos_L[bcidx].bcDepth_dt_m_tp1;
			}
		}
		if (dnp1 < 0) { dnp1 = 0; }
		double resd = dnp1 - dn;
		cvs_L[i].dp_tp1 = dnp1;
		cvs_L[i].hp_tp1 = cvs_L[i].dp_tp1 + cvsele_L[i];
		if (abs(resd) <= CCh) { break; }
	}
	cvs_L[i].resd = abs(cvs_L[i].dp_tp1 - dp_old);
	if (cvs_L[i].resd > CCh) {
		return 0;
	}
	return 1;
}


__host__ __device__ void calWFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx){
	if (gvi_L.nCols == 1) { return; }
	fluxData flxw; //W, x-
	if (cvs_L[idx].colx == 0 || cvs_L[idx].cvidx_atW == -1)//w 측 경계셀
	{
		if (cvs_L[idx].isBCcell == 1) {
			flxw = noFlx(); // w측 최 경계에서는 w 방향으로 flx 없다.
		}
		else {// w측 최 경계에서는 w 방향으로 자유수면 flx 있다.
			double slp_tm1 = 0;
			if (cvs_L[idx].cvdix_atE >= 0)
			{
				double he = cvs_L[cvs_L[idx].cvdix_atE].dp_t + cvsele_L[cvs_L[idx].cvdix_atE];
				double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
				slp_tm1 = (he - hcur) / gvi_L.dx; //i+1 셀과의 e 수면경사를 w 방향에 적용한다.
			}
			//double slp_tm1 = (cvs[cvs[idx].cvaryNum_atE].hp_t - cvs[idx].hp_t) / gv.dx; //i+1 셀과의 수면경사를 w 방향에 적용한다.
			slp_tm1 = slp_tm1 + gvi_L.domainOutBedSlope;
			if (slp_tm1 >= slpMIN && cvs_L[idx].dp_tp1 > dMinLimit)
			{
				flxw = calMEq_DWEm_Deterministric(cvs_L[idx].qw_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0);
			}
			else { flxw = noFlx(); }
		}
	}
	else {
		if (cvs_L[idx].isSimulatingCell == 0) {
			flxw = noFlx();
		}
		else {
			flxw.v = cvs_L[cvs_L[idx].cvidx_atW].ve_tp1;
			flxw.slp = cvs_L[cvs_L[idx].cvidx_atW].slpe;
			flxw.q = cvs_L[cvs_L[idx].cvidx_atW].qe_tp1;
			flxw.dflow = cvs_L[cvs_L[idx].cvidx_atW].dfe;
		}
	}
	cvs_L[idx].qw_tp1 = flxw.q;
	//cvs[idx].vw_tp1 = flxw.v;
	//cvs[idx].dfw = flxw.dflow;
}

__host__ __device__ void calEFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx){
	if (gvi_L.nCols == 1) { return; }
	fluxData flxe;    //E,  x+
	if (cvs_L[idx].colx == (gvi_L.nCols - 1) || cvs_L[idx].cvdix_atE == -1) {
		if (cvs_L[idx].isBCcell == 1) { flxe = noFlx(); }
		else {
			double slp_tm1 = 0;
			if (cvs_L[idx].cvidx_atW >= 0) {
				//double slp = (cell.hp_tp1 - dm.cells[cx - 1, ry].hp_tp1) / dx; //i-1 셀과의 수면경사를 e 방향에 적용한다.
				double hw = cvs_L[cvs_L[idx].cvidx_atW].dp_t + cvsele_L[cvs_L[idx].cvidx_atW];
				double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
				slp_tm1 = (hcur - hw) / gvi_L.dx;
			}
			//double slp_tm1 = (cvs[idx].hp_t - cvs[cvs[idx].cvaryNum_atW].hp_t) / gv.dx;
			slp_tm1 = slp_tm1 - gvi_L.domainOutBedSlope;
			if (slp_tm1 <= (-1 * slpMIN) && cvs_L[idx].dp_tp1 > dMinLimit) {
				flxe = calMEq_DWEm_Deterministric(cvs_L[idx].qe_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0);
			}
			else { flxe = noFlx(); }
		}
	}
	else {
		if (cvs_L[idx].isSimulatingCell == 0) {
			flxe = noFlx();
		}
		else {
			//flxe = getFluxToEorS(cvs[idx], cvs[cvs[idx].cvdix_atE], 1);
			flxe = getFluxToEorS(cvs_L, cvsele_L, gvi_L, idx, cvs_L[idx].cvdix_atE, 1); 
		}
	}
	cvs_L[idx].ve_tp1 = flxe.v;
	cvs_L[idx].dfe = flxe.dflow;
	cvs_L[idx].slpe = flxe.slp;
	cvs_L[idx].qe_tp1 = flxe.q;
}


__host__ __device__ void calNFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx) {
	if (gvi_L.nRows == 1) { return; }
	fluxData flxn;  //N, y-
	if (cvs_L[idx].rowy == 0 || cvs_L[idx].cvidx_atN == -1) {
		if (cvs_L[idx].isBCcell == 1) { flxn = noFlx(); }
		else {// n측 최 경계에서는 n 방향으로 자유수면 flx 있다.
			double slp_tm1 = 0;
			if (cvs_L[idx].cvidx_atS >= 0) {
				//double slp = (dm.cells[cx, ry + 1].hp_tp1 - cell.hp_tp1) / dx; //j+1 셀과의 수면경사를 w 방향에 적용한다.
				//double slp_tm1 = (cvs[cvs[idx].cvaryNum_atS].hp_t - cvs[idx].hp_t) / gv.dx; //j+1 셀과의 수면경사를 w 방향에 적용한다.
				double hs = cvs_L[cvs_L[idx].cvidx_atS].dp_t + cvsele_L[cvs_L[idx].cvidx_atS];
				double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
				slp_tm1 = (hs - hcur) / gvi_L.dx;
			}
			slp_tm1 = slp_tm1 + gvi_L.domainOutBedSlope;
			if (slp_tm1 >= slpMIN
				&& cvs_L[idx].dp_tp1 > dMinLimit) {
				//flxn = getFluxToDomainOut(cell, slp_tm1, cell.qn_t, cell.vn_t, gv.gravity, dt_sec);
				flxn = calMEq_DWEm_Deterministric(cvs_L[idx].qn_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0);
			}
			else { flxn = noFlx(); }
		}
	}
	else {
		if (cvs_L[idx].isSimulatingCell == 0) {
			flxn = noFlx();
		}
		else {
			flxn.v = cvs_L[cvs_L[idx].cvidx_atN].vs_tp1;
			flxn.slp = cvs_L[cvs_L[idx].cvidx_atN].slps;
			flxn.dflow = cvs_L[cvs_L[idx].cvidx_atN].dfs;
			flxn.q = cvs_L[cvs_L[idx].cvidx_atN].qs_tp1;
		}
	}
	cvs_L[idx].qn_tp1 = flxn.q;
	//cvs[idx].vn_tp1 = flxn.v;
	//cvs[idx].dfn = flxn.dflow;
}


__host__ __device__ void calSFlux(cvatt* cvs_L, double* cvsele_L, globalVinner gvi_L, int idx) {
	if (gvi_L.nRows == 1) { return; }
	fluxData flxs;//S, y+
	if (cvs_L[idx].rowy == (gvi_L.nRows - 1)
		|| cvs_L[idx].cvidx_atS == -1) {
		if (cvs_L[idx].isBCcell == 1) { flxs = noFlx(); }
		else {
			double slp_tm1 = 0;
			if (cvs_L[idx].cvidx_atN >= 0) {
				//double slp = (cell.hp_tp1 - dm.cells[cx, ry - 1].hp_tp1) / dx; //i-1 셀과의 수면경사를 e 방향에 적용한다.
				//double slp_tm1 = (cvs[idx].hp_t - cvs[cvs[idx].cvaryNum_atN].hp_t) / gv.dx; //i-1 셀과의 수면경사를 e 방향에 적용한다.
				double hn = cvs_L[cvs_L[idx].cvidx_atN].dp_t + cvsele_L[cvs_L[idx].cvidx_atN];
				double hcur = cvs_L[idx].dp_t + cvsele_L[idx];
				slp_tm1 = (hcur - hn) / gvi_L.dx;
			}
			slp_tm1 = slp_tm1 - gvi_L.domainOutBedSlope;
			if (slp_tm1 <= (-1 * slpMIN)
				&& cvs_L[idx].dp_tp1 > dMinLimit) {
				//flxs = getFluxToDomainOut(cell, slp_tm1, cell.qs_t, cell.vs_t, gv.gravity, dt_sec);
				flxs = calMEq_DWEm_Deterministric(cvs_L[idx].qs_t,
					gvi_L.dt_sec, slp_tm1, cvs_L[idx].rc, cvs_L[idx].dp_tp1, 0);
			}
			else { flxs = noFlx(); }
		}
	}
	else {
		if (cvs_L[idx].isSimulatingCell == 0) {
			flxs = noFlx();
		}
		else {
			//flxs = getFluxToEorS(cvs[idx], cvs[cvs[idx].cvidx_atS], 3);
			flxs = getFluxToEorS(cvs_L, cvsele_L, gvi_L, idx, cvs_L[idx].cvidx_atS, 3);
		}
	}
	cvs_L[idx].vs_tp1 = flxs.v;
	cvs_L[idx].dfs = flxs.dflow;
	cvs_L[idx].slps = flxs.slp;
	cvs_L[idx].qs_tp1 = flxs.q;
}

__host__ __device__ fluxData noFlx(){
	fluxData flx;
	flx.dflow = 0;
	flx.fd = 0;
	flx.q = 0;
	flx.slp = 0;
	flx.v = 0;
	return flx;
}

__host__ __device__ fluxData calMEq_DWEm_Deterministric(double qt, 
	double dt_sec, double slp, double rc, double dflow, double qt_ip1){
	fluxData flx;
	double qapp = qt;
	//double q = (qapp - (gravity * dflow * dt_sec * slp)) /
	//         (1 + gravity * dt_sec * (rc * rc) * DeviceFunction.Sqrt((qapp * qapp + qt_ip1 * qt_ip1) / 2) 
	//         / DeviceFunction.Pow(dflow, (double)7 / 3));
	//double q = (qapp - (GRAVITY * dflow * dt_sec * slp)) /
	//	(1 + GRAVITY * dt_sec * (rc * rc) * abs(qapp) / pow(dflow, 7.0 / 3.0));
	double term2 = GRAVITY * dflow * dt_sec * slp;
	double term3 = GRAVITY * dt_sec * (rc * rc) * abs(qapp);
	double term4 = pow(dflow, 7.0 / 3.0);
	double q = (qapp - term2) / (1 + term3 / term4);

	flx.q = q;
	flx.v = flx.q / dflow;  // Manning 결과와 같다. flx.v = Math.Pow(dflow, 2 / 3) * Math.Abs(slp) / mN; 
	flx.dflow = dflow;
	flx.slp = slp;
	return flx; ;
}

//targetCellDir : N = 1, E = 4, S = 16, W = 64, NONE = 0
__host__ __device__ fluxData getFluxToEorS(cvatt* cvs_L, double* cvsele_L,
	globalVinner gvi_L, int idxc, int idxt, int targetCellDir)
{
	cvatt curCell = cvs_L[idxc];
	cvatt tarCell = cvs_L[idxt];
	double slp = 0;
	//double dht = (tarCell.elez+tarCell .dp_t)-(curCell.elez+curCell.dp_t); //+면 자신의 셀이, 대상 셀보다 낮다, q는 -, slp는 +.   -면 자신의 셀이, 대상 셀보다 높다, q는 +, slp는 - 
	double dhtp1 = tarCell.hp_tp1 - curCell.hp_tp1;
	if (dhtp1 == 0) { return noFlx(); }
	if (dhtp1 > 0
		&& tarCell.dp_tp1 <=dMinLimit) {/*		gvi_L.dMinLimitforWet*/
		return noFlx();
	}
	if (dhtp1 < 0
		&& curCell.dp_tp1 <= dMinLimit) {
		return noFlx();
	}
	slp = dhtp1 / gvi_L.dx;
	if (abs(slp) < slpMIN	|| slp == 0) {
		return noFlx();
	}
	double dflow = max(curCell.hp_tp1, tarCell.hp_tp1)
		- max(cvsele_L[idxc], cvsele_L[idxt]);
	// 최대 수심법
	//dflow = DeviceFunction.Max(curCell.hp_tp1, tarCell.hp_tp1); 
	//// 수심평균 법
	//double maxBedElev = DeviceFunction.Max(curCell.elez, tarCell.elez);
	//double d1 = curCell.hp_tp1 - maxBedElev;
	//if (d1 < 0) { d1 = 0; }
	//double d2 = tarCell.hp_tp1 - maxBedElev;
	//if (d2 < 0) { d2 = 0; }
	//dflow = (d1 + d2) / 2;
	//// 수심평균 법
	if (dflow <= 0) { return noFlx(); }
	double qt = 0; 
	double qtp1 = 0;
	double q_ip1 = 0;
	double u_ip1 = 0;
	if (targetCellDir == 1) {
		qt = curCell.qe_t;
		qtp1 = curCell.qe_tp1; // qtp1
		u_ip1 = tarCell.ve_tp1; q_ip1 = tarCell.qe_tp1;
	}
	else if (targetCellDir == 3) {
		qt = curCell.qs_t;
		qtp1 = curCell.qs_tp1;
		u_ip1 = tarCell.vs_tp1; q_ip1 = tarCell.qs_tp1;
	}
	fluxData flx;
	if (isDWE) {
		//flx = calFluxUsingME_DWE_Implicit_UsingGPU(dhtp1, qt, qtp1, dflow, currentCell.rc, dx, dt_sec);
		flx = calMEq_DWE_Deterministric(qt, dflow,
			slp, curCell.rc, gvi_L.dx, gvi_L.dt_sec, q_ip1, u_ip1);
	}
	else {
		//flx = calFluxUsingME_mDWE_Implicit(dhtp1, dht,
		//       qt, qtp1, dflow, currentCell.lc.roughnessCoeff, dx, dt_sec, currentCell.colxary, currentCell.rowyary);
		flx = calMEq_DWEm_Deterministric(qt,
			gvi_L.dt_sec, slp, curCell.rc, dflow, q_ip1);
	}
	if (!isAS) {
		if (abs(flx.q) > 0) {
			flx = getFluxUsingSubCriticalCon(flx, gvi_L.froudeNCriteria);
			flx = getFluxUsingFluxLimit(flx, dflow, gvi_L.dx, gvi_L.dt_sec);
			//flx = getFluxqUsingFourDirLimitUsingDepthCondition(currentCell, flx, dflow, dx, dt_sec); //이건 수렴이 잘 안된다.
			//flx = getFluxUsingFourDirLimitUsingCellDepth(currentCell, targetCell, flx, dx, dt_sec);
			//flx = getFluxUsingFourDirLimitUsingDh(flx, dhtp1, dx, dt_sec); // 이건 소스에서 수심이 급격히 올라간다.
		}
	}
	flx.slp = slp;
	return flx;
}


__host__ __device__ fluxData calMEq_DWE_Deterministric(double qt, double dflow,
	double slp, double rc, float dx, double dt_sec, double q_ip1, double u_ip1)
{
	// 이거 잘 안된다. 반복법이 필요.. 2018.12.26.
	fluxData flx;
	double qapp = qt; //Math.Abs(qt);
	//2019.1.2 관성이 없을 경우에는 
	// slp가 + 면 q는 -, slp가 - 이면 q는 + 가 되어야 함.
	// 이전 t에서 q 가  0 이면, slp가 + 일때 무조건 q는 - , slp가 - 일때는 q는 무조건 +.
	// 이전 t에서 q 가  - 이면, slp가 + 일때 무조건 q는 - , slp가 - 일때는 q는 - 일수도 있고, + 일수도 있음. => 조건 처리 필요
	// 이전 t에서 q 가 + 이면, slp가 + 일때 q는 - 일수도 있고, + 일수도 있음, slp가 - 일때는 q는 무조건 +. => 조건 처리 필요

	double ut = qapp / dflow;
	double q = (qapp - (GRAVITY * dflow * dt_sec * slp)) /
		(1 + ut * dt_sec / dx + GRAVITY * dt_sec * (rc * rc) * abs(qapp) / pow(dflow, 7.0 / 3.0));
	//double q = ((qapp - q_ip1 * u_ip1 * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//                (1 - ut * dt_sec / dx + gravity * dt_sec * (rc * rc) * DeviceFunction.Abs(qapp) 
	//                / DeviceFunction.Pow(dflow, (double)7 / 3)));
	//double q = ((qapp - Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2) * (u_ip1+ut)/2 * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//              (1 - (u_ip1 + ut) / 2 * dt_sec / dx + gravity * dt_sec * (rc * rc) * Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2) 
	//             / DeviceFunction.Pow(dflow, (double)7 / 3)));
	//double q = ((qapp - Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2) * (u_ip1 + ut) / 2 * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//               (1 - ut * dt_sec / dx + gravity * dt_sec * (rc * rc) * Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2) 
	//              / DeviceFunction.Pow(dflow, (double)7 / 3)));
	//double q = ((qapp - Math.Sqrt((q_ip1 * q_ip1 + qapp * qapp) / 2) * ut * dt_sec / dx - (gravity * dflow * dt_sec * slp)) /
	//   (1 - ut * dt_sec / dx + gravity * dt_sec * (rc * rc) * qapp / DeviceFunction.Pow(dflow, (double)7 / 3)));

	flx.q = q;
	flx.v = flx.q / dflow;  // Manning 결과와 같다. flx.v = Math.Pow(dflow, 2 / 3) * Math.Abs(slp) / mN; 
	flx.dflow = dflow;
	flx.slp = slp;
	return flx; ;
}

__host__ __device__ fluxData getFluxUsingSubCriticalCon(fluxData inflx, float froudNCriteria){
	double v_wave = sqrt(GRAVITY * inflx.dflow);
	double fn = abs(inflx.v) / v_wave;
	double qbak = inflx.q;
	if (fn > froudNCriteria)
	{
		double v = froudNCriteria * v_wave;
		inflx.v = v;
		if (qbak < 0) { inflx.v = -1 * v; }
		inflx.q = inflx.v * inflx.dflow;
	}
	return inflx;
}

__host__ __device__ fluxData getFluxUsingFluxLimit(fluxData inflx, double dflow, 
	float dx, double dt_sec){
	double qmax = abs(dflow) * dx / 2 / dt_sec; // 수위차의 1/2 이 아니라, 흐름 수심의 1/2이므로, 수위 역전 될 수 있다.
	double qbak = inflx.q;
	if (abs(inflx.q) > qmax) {
		inflx.q = qmax;
		if (qbak < 0) { inflx.q = -1 * qmax; }
		inflx.v = inflx.q / inflx.dflow;
	}
	return inflx;
}

__host__ __device__ void initializeThisStepAcell(cvatt* cvs_L, cvattAddAtt* cvsAA_L,
	bcAppinfo* bcAppinfos_L, double elev, double rfi_read_mPs_L,
	int idx, thisProcessInner psi_L, globalVinner gvi_L)
{
	double h = cvs_L[idx].dp_tp1 + elev;// cvs_L[idx].elez; //elev 가 변경되는 경우가 있으므로, 이렇게 수위설정
	if (cvs_L[idx].hp_tp1 <= h) { // 지면고가 높아진 경우
		// dem  고도 변경되면, 수심이 바뀐다. 수위는 유지.
		// cvs_L[idx].hp_t=cvs_L[idx].elez + cvs_L[idx].dp_t 이므로, cvs_L[idx].dp_t 이값과 cvs_L[idx].dp_tp1  모두 업데이트 해줘야 한다.
		cvs_L[idx].dp_tp1 = cvs_L[idx].hp_tp1 - elev;
		if (cvs_L[idx].dp_tp1 < 0) { cvs_L[idx].dp_tp1 = 0; }
		cvs_L[idx].dp_t = cvs_L[idx].dp_tp1;
	}
	else {
		cvs_L[idx].dp_t = cvs_L[idx].dp_tp1;
	}
	cvs_L[idx].qe_t = cvs_L[idx].qe_tp1;
	cvs_L[idx].qw_t = cvs_L[idx].qw_tp1;
	cvs_L[idx].qs_t = cvs_L[idx].qs_tp1;
	cvs_L[idx].qn_t = cvs_L[idx].qn_tp1;
	double sourceAlltoRoute_tp1_dt_m = 0.0;
	if (cvs_L[idx].isBCcell == 1) { // prj.isbcApplied == 1 조건은 보장됨
		int bci = getBcAppinfoidx(bcAppinfos_L, gvi_L.bcCellCountAll, idx);
		bcAppinfos_L[bci].bcDepth_dt_m_tp1 = getCDasDepthWithLinear(bcAppinfos_L[bci].bctype,
			bcAppinfos_L[bci].bcData_curOrder, bcAppinfos_L[bci].bcData_nextOrder,
			bcAppinfos_L[bci].bcData_curOrderStartedTime_sec, elev, psi_L, gvi_L);

		if (bcAppinfos_L[bci].bctype == 1)//1:  Discharge,  2: Depth, 3: Height,  4: None
		{//경계조건이 유량일 경우, 소스항에 넣어서 홍수추적한다. 수심으로 환산된 유량..
			sourceAlltoRoute_tp1_dt_m = bcAppinfos_L[bci].bcDepth_dt_m_tp1;
		}
		else
		{//경계조건이 유량이 아닐경우, 홍수추적 하지 않고, 고정된 값 적용.
			cvs_L[idx].dp_tp1 = bcAppinfos_L[bci].bcDepth_dt_m_tp1;
			if (psi_L.tnow_sec == 0) {
				cvs_L[idx].dp_t = cvs_L[idx].dp_tp1;
			}
		}
	}
	cvsAA_L[idx].sourceRFapp_dt_meter = 0;
	//-1, 0 :false, 1: true
	if (gvi_L.isRFApplied == 1 && psi_L.rfEnded == 0)
	{
		if (gvi_L.rfType == rainfallDataType::TextFileASCgrid) {
			cvsAA_L[idx].sourceRFapp_dt_meter = rfi_read_mPs_L * gvi_L.dt_sec;
		}
		else {
			cvsAA_L[idx].sourceRFapp_dt_meter = psi_L.rfReadintensityForMAP_mPsec * gvi_L.dt_sec;
		}
	}
	sourceAlltoRoute_tp1_dt_m = sourceAlltoRoute_tp1_dt_m + cvsAA_L[idx].sourceRFapp_dt_meter;
	cvs_L[idx].dp_t = cvs_L[idx].dp_t + sourceAlltoRoute_tp1_dt_m;
	cvs_L[idx].dp_tp1 = cvs_L[idx].dp_tp1 + sourceAlltoRoute_tp1_dt_m;
	cvs_L[idx].hp_tp1 = cvs_L[idx].dp_tp1 + elev;
	//if (cvs_L[idx].dp_tp1 > gvi_L.dMinLimitforWet) {
	//	setEffCells(cvs_L, idx);
	//}
}

__host__ __device__ void setStartingConditionCVs_inner(cvatt* cvs_L, cvattAddAtt* cvsAA_L,
	double* cvselez_L, int idx) {
	cvs_L[idx].dp_t = cvsAA_L[idx].initialConditionDepth_m;
	cvs_L[idx].dp_tp1 = cvs_L[idx].dp_t;
	cvs_L[idx].ve_tp1 = 0;
	cvs_L[idx].qe_tp1 = 0;
	cvs_L[idx].qw_tp1 = 0;
	cvs_L[idx].qn_tp1 = 0;
	cvs_L[idx].qs_tp1 = 0;
	//cvs_L[idx].hp_tp1 = cvs_L[idx].dp_tp1 + cvs_L[idx].elez;
	cvs_L[idx].hp_tp1 = cvs_L[idx].dp_tp1 + cvselez_L[idx];
	cvsAA_L[idx].fdmaxV = 0;//E = 1, S = 3, W = 5, N = 7, NONE = 0
	//cvsAA_L[idx].bcData_curOrder = 0;
	cvsAA_L[idx].sourceRFapp_dt_meter = 0;
	//cvsAA_L[idx].rfReadintensity_mPsec = 0;
	cvs_L[idx].isSimulatingCell = 0;
}


__host__ __device__ double getCDasDepthWithLinear(int bctype, double vcurOrder, double vnextOrder,
	int t_curOrderStarted_sec, double elev_m, thisProcessInner psi_L, globalVinner gvi_L)
{
	double valueAsDepth_curOrder = 0;
	double valueAsDepth_nextOrder = 0;
	double dx = gvi_L.dx;
	double dt_s = gvi_L.dt_sec;
	//1:  Discharge,  2: Depth, 3: Height,  4: None
	switch (bctype)
	{
	case 1://conditionDataType::Discharge:
		valueAsDepth_curOrder = (vcurOrder / dx / dx) * dt_s;
		valueAsDepth_nextOrder = (vnextOrder / dx / dx) * dt_s;
		break;
	case 2://conditionDataType::Depth:
		valueAsDepth_curOrder = vcurOrder;
		valueAsDepth_nextOrder = vnextOrder;
		break;
	case 3://conditionDataType::Height:
		valueAsDepth_curOrder = vcurOrder - elev_m;
		valueAsDepth_nextOrder = vnextOrder - elev_m;
		break;
	}
	if (valueAsDepth_curOrder < 0) { valueAsDepth_curOrder = 0; }
	if (valueAsDepth_nextOrder < 0) { valueAsDepth_nextOrder = 0; }
	double bcDepth_dt_m_tp1 = 0.0;
	if (!isAS) { // 해석해 테스트가 아닐때는 이 조건 사용
		bcDepth_dt_m_tp1 = (valueAsDepth_nextOrder - valueAsDepth_curOrder)
			* (psi_L.tnow_sec - t_curOrderStarted_sec) / gvi_L.dtbc_sec
			+ valueAsDepth_curOrder;
	}
	else {
		bcDepth_dt_m_tp1 = valueAsDepth_curOrder; // 해석해 테스트는 이 조건
	}
	return  bcDepth_dt_m_tp1;
}

__host__ __device__ int getBcAppinfoidx(bcAppinfo * bcAppinfos, int bcCellCountAll, int cvidxToGet) {

	for (int i = 0; i < bcCellCountAll; ++i) {
		if (bcAppinfos[i].cvidx == cvidxToGet) {
			return i;
		}
	}
	return -1;
}

__host__ __device__ void setEffCells(cvatt * cvs_L, int idx)
{
	cvs_L[idx].isSimulatingCell = 1;
	if (cvs_L[idx].cvdix_atE >= 0) {
		cvs_L[cvs_L[idx].cvdix_atE].isSimulatingCell = 1;
	}
	if (cvs_L[idx].cvidx_atW >= 0) {
		cvs_L[cvs_L[idx].cvidx_atW].isSimulatingCell = 1;
	}
	if (cvs_L[idx].cvidx_atN >= 0) {
		cvs_L[cvs_L[idx].cvidx_atN].isSimulatingCell = 1;
	}
	if (cvs_L[idx].cvidx_atS >= 0) {
		cvs_L[cvs_L[idx].cvidx_atS].isSimulatingCell = 1;
	}
}



__host__ __device__ fluxData getFD4MaxValues(cvatt * cvs_L, int i) {
	fluxData flxmax;
	if (cvs_L[i].cvidx_atW >= 0 && cvs_L[i].cvidx_atN >= 0) {
		//  이경우는 4개 방향 성분에서 max 값 얻고
		flxmax = getFD4MaxValues_inner(cvs_L, i,
			cvs_L[i].cvidx_atW,
			cvs_L[i].cvidx_atN);
	}
	else if (cvs_L[i].cvidx_atW >= 0 && cvs_L[i].cvidx_atN < 0) {
		flxmax = getFD4MaxValues_inner(cvs_L, i,
			cvs_L[i].cvidx_atW, i);
	}
	else  if (cvs_L[i].cvidx_atW < 0 && cvs_L[i].cvidx_atN >= 0) {
		flxmax = getFD4MaxValues_inner(cvs_L, i,
			i, cvs_L[i].cvidx_atN);
	}
	else {//w, n에 셀이 없는 경우
		flxmax = getFD4MaxValues_inner(cvs_L, i, i, i);
	}
	return flxmax;
}

__host__ __device__ fluxData getFD4MaxValues_inner(cvatt* cvs_L, int ip, int iw, int in)
{	// cell을 전달 받는 것 보다, index를 받아서 지역변수로 cell을 선언하는게 더 빠르다..2020.05.12
	fluxData flxmax;
	cvatt wcell = cvs_L[iw];
	cvatt cell = cvs_L[ip];
	cvatt ncell = cvs_L[in];
	double vw = abs(wcell.ve_tp1);
	double ve = abs(cell.ve_tp1);
	double vn = abs(ncell.vs_tp1);
	double vs = abs(cell.vs_tp1);
	double vmaxX = max(vw, ve);
	double vmaxY = max(vn, vs);
	double vmax = max(vmaxX, vmaxY);
	if (vmax == 0) {
		flxmax.fd = 0;// cVars.FlowDirection4.NONE;
		flxmax.v = 0;
		flxmax.dflow = 0;
		flxmax.q = 0;
		return flxmax;
	}
	else {
		flxmax.v = vmax;//E = 1, S = 3, W = 5, N = 7, NONE = 0
		if (vmax == vw) {
			flxmax.fd = 5;
		}
		else if (vmax == ve) {
			flxmax.fd = 1;
		}
		else if (vmax == vn) {
			flxmax.fd = 7;
		}
		else if (vmax == vs) {
			flxmax.fd = 3;
		}
	}
	double dmaxX = max(wcell.dfe, cell.dfe);
	double dmaxY = max(ncell.dfs, cell.dfs);
	flxmax.dflow = max(dmaxX, dmaxY);
	double qw = abs(wcell.qe_tp1);
	double qe = abs(cell.qe_tp1);
	double qn = abs(ncell.qs_tp1);
	double qs = abs(cell.qs_tp1);
	double qmaxX = max(qw, qe);
	double qmaxY = max(qn, qs);
	flxmax.q = max(qmaxX, qmaxY);
	return flxmax;
}


__host__ __device__ double getVNConditionValue(cvatt* cvs_L, int i) {
	double searchMIN = DBL_MAX;
	double curValue = 0;
	double rc = cvs_L[i].rc;
	// e 값과 중복되므로, w는 계산하지 않는다.
	if (cvs_L[i].dfe > 0) {
		searchMIN = 2 * rc * sqrt(abs(cvs_L[i].slpe))
			/ pow(cvs_L[i].dfe, 5.0 / 3.0);
		//if (curValue < searchMIN) {
		//	searchMIN = curValue;
		//}
	}
	// s 값과 중복되므로, n는 계산하지 않는다.
	if (cvs_L[i].dfs > 0) {
		curValue = 2 * rc * sqrt(abs(cvs_L[i].slps))
			/ pow(cvs_L[i].dfs, 5.0 / 3.0);
		if (curValue < searchMIN) {
			searchMIN = curValue;
		}
	}
	return searchMIN;
}






