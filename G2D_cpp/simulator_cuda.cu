#include "hip/hip_runtime.h"
﻿
#pragma once

#ifdef __INTELLISENSE___
void __syncthreads();
#else
void __syncthreads();
#endif

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include "" // cuda에서 정의된 키워드 포함
#include <float.h>

#include "g2d_cuda.cuh"
#include "g2d.h"

extern projectFile prj;
extern domaininfo di;
extern generalEnv ge;

extern cvatt* cvs;
extern cvattAdd* cvsAA;

extern thisProcess ps;
extern thisProcessInner psi;
extern globalVinner gvi[1];

int simulationControl_GPU()
{
	hipDeviceProp_t m_deviceProp;
	hipGetDeviceProperties(&m_deviceProp, 0);
	if (!m_deviceProp.canMapHostMemory) {
		fprintf(stderr, "Device does not support mapping CPU host memory!\n");
		return 0;
	}
	int	nRows = di.nRows;
	int nCols = di.nCols;
	double dx = di.dx;
	double simDur_min = prj.simDuration_min + 1.0;
	int bcDataOrder = 0;
	int rfDataOrder = 0;
	int demToChangeEnded = 1;
	double tnow_min_bak = 0;
	ps.thisPrintStepStartTime = COleDateTime::GetCurrentTime();
	ps.simulationStartTime = COleDateTime::GetCurrentTime();
	ps.rfEnded = 1;
	ps.tsec_targetToprint = 0;
	ps.dt_printout_sec = (int)(prj.printOutInterval_min * 60);
	ps.tnow_sec = 0;
	ps.effCellCount = 0;
	ps.dtbc_sec = prj.bcDataInterval_min * 60;
	ps.tTag_length = prj.tTag_length;
	int dtbc_min = prj.bcDataInterval_min;
	// gpu에서 구조체 변수 업데이트 안됨 (c#). 그래서 구조체 배열로 사용. 
	// C++에서 포인터 혹은 구조체 로 전환시 gpu 코딩에서 변수 업데이트 되는지 확인 필요
	int onCPU = 1;
	if (prj.isRainfallApplied == 1) {
		ps.rfEnded = 0;
	}
	if (prj.isDEMtoChangeApplied == 1) {
		demToChangeEnded = 0;
	}
	gvi[0] = initGlobalVinner();
	psi.dt_sec = ge.dtStart_sec;

	size_t  ms_cvs = di.cellNnotNull * sizeof(cvatt);
	size_t  ms_cvsAdd = di.cellNnotNull * sizeof(cvattAdd);
	cvatt* d_cvs;
	cvattAdd* d_cvsAA;

	hipMalloc((void**)& d_cvs, ms_cvs);
	hipMalloc((void**)& d_cvsAA, ms_cvsAdd);
	hipMemcpy(d_cvs, cvs, ms_cvs, hipMemcpyHostToDevice);
	hipMemcpy(d_cvsAA, d_cvsAA, ms_cvsAdd, hipMemcpyHostToDevice);
	dim3 threadsPerBlock(512, 1, 1); //blockDim    
	dim3 blocksPerGrid(di.cellNnotNull / (threadsPerBlock.x * threadsPerBlock.y) + 1, 1); //gridDim 


	setStartingCondition_GPU(d_cvs, d_cvsAA, di.cellNnotNull, blocksPerGrid, threadsPerBlock);
	do { //모의 시작할 때 t 는 초기 조건, t+dt는 소스 하나가 적용된 결과
		ps.tnow_min = ps.tnow_sec / 60.0;
		if (prj.isbcApplied == 1) {//경계조건 등
			int bc_min = bcDataOrder * dtbc_min;
			if (((tnow_min_bak < bc_min) & (ps.tnow_min >= bc_min))
				|| bc_min == 0) {
				bcDataOrder++;
				getCellCD(bcDataOrder, dtbc_min);
			}
		}
		if (prj.isRainfallApplied == 1 && ps.rfEnded == 0) {//강우
			int rf_min = rfDataOrder * prj.rainfallDataInterval_min;
			if (((tnow_min_bak < rf_min) & (ps.tnow_min >= rf_min))
				|| rf_min == 0) {
				psi.rfisGreaterThanZero = 0;
				rfDataOrder++; //1부터 시작. 배열은 rainfallDataOrder-1
				ps.rfEnded = readRainfallAndGetIntensity(rfDataOrder);
				// 0보다 큰 강우가 하나라도 있으면...
				if (psi.rfisGreaterThanZero == 1) {
					gvi[0].dMinLimitforWet = ge.dMinLimitforWet_ori;
				}
				else if (psi.rfisGreaterThanZero == 0 || ps.rfEnded == 1) {
					//강우가 없을때는 최소수심을 좀 크게 잡아도 된다.
					gvi[0].dMinLimitforWet = ge.dMinLimitforWet_ori * 5.0;
				}
			}
		}
		if (prj.isDEMtoChangeApplied == 1 && demToChangeEnded == 0) {//dem file 교체
			demToChangeEnded = changeDomainElevWithDEMFile(ps.tnow_min, tnow_min_bak);
		}
		initilizeThisStep_GPU << < blocksPerGrid, threadsPerBlock >> > (d_cvs, di.cellNnotNull);

		runSolver_GPU();
		updateValuesInThisStepResults();
		if (ps.tnow_sec >= ps.tsec_targetToprint) {
			checkEffCellNandSetAllFalse();// 출력할때 마다 이 정보 업데이트
			makeOutputFiles(ps.tnow_sec);
			int progressRatio = (int)(ps.tnow_min / prj.simDuration_min * 100);
			printf("\rCurrent progress[min]: %d/%d[%d%%]..", (int)ps.tnow_min,
				(int)prj.simDuration_min, progressRatio);
			//한번 출력할때 마다 모의변수 업데이트
			if (updateProjectParameters() == 0) {
				return 0;
			}
			ps.tsec_targetToprint = ps.tsec_targetToprint + ps.dt_printout_sec;
			ps.thisPrintStepStartTime = COleDateTime::GetCurrentTime();
		}
		tnow_min_bak = ps.tnow_min;
		ps.tnow_sec = ps.tnow_sec + psi.dt_sec;
		if (prj.isFixedDT == 0) {
			psi.dt_sec = getDTsecWithConstraints(psi.dflowmaxInThisStep,
				psi.vmaxInThisStep, psi.VNConMinInThisStep);
		}
	} while (ps.tnow_min < simDur_min);
	return 1;
}

__global__ void initilizeThisStep_GPU(cvatt* cvs_k, int arraySize)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	// Check if thread is within array bounds. threadid 보다 큰 array는 계산이 안된다.
	if (tid < arraySize) {
		initializeThisStepAcell(tid);
	}
}


void setStartingCondition_GPU(cvatt* cvs_k, cvattAdd* cvsadd_k, 
	int arraySize, dim3 blocksPerGrid, dim3 threadsPerBlock)
{
	ps.floodingCellDepthThresholds_m.clear();
	if (prj.floodingCellDepthThresholds_cm.size() < 1) {
		ps.floodingCellDepthThresholds_m.push_back(ge.dMinLimitforWet_ori);
	}
	else {
		for (int i = 0; i < prj.floodingCellDepthThresholds_cm.size(); ++i) {
			double v = prj.floodingCellDepthThresholds_cm[i] / 100.0;
			ps.floodingCellDepthThresholds_m.push_back(v);
		}
	}
	setStartingConditionCVs_GPU << < blocksPerGrid, threadsPerBlock >> > (cvs_k, arraySize);
}

__global__ void setStartingConditionCVs_GPU(cvatt* cvs_k, cvattAdd* cvsadd_k,
	int arraySize) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < arraySize) {
		cvs_k[tid].dp_t = cvsadd_k[tid].initialConditionDepth_m;
		cvs_k[tid].dp_tp1 = cvs_k[tid].dp_t;
		cvs_k[tid].ve_tp1 = 0;
		cvs_k[tid].qe_tp1 = 0;
		cvs_k[tid].qw_tp1 = 0;
		cvs_k[tid].qn_tp1 = 0;
		cvs_k[tid].qs_tp1 = 0;
		cvs_k[tid].hp_tp1 = cvs_k[tid].dp_tp1 + cvs_k[tid].elez;
		cvsadd_k[tid].fdmax = 0;// N = 1, E = 4, S = 16, W = 64, NONE = 0
		cvsadd_k[tid].bcData_curOrder = 0;
		cvsadd_k[tid].sourceRFapp_dt_meter = 0;
		cvsadd_k[tid].rfReadintensity_mPsec = 0;
		cvs_k[tid].isSimulatingCell = 0;
	}
}


void runSolver_GPU(cvatt* cvs_k, int arraySize){
	int nCells = gvi[0].nCellsInnerDomain;
	int thdWet = gvi[0].dMinLimitforWet;
	// 여기서는 배열, critical 속도 같다...
	psi.iGSmax = 0;
//	for (int igs = 0; igs < gvi[0].iGSmaxLimit; igs++) {
//		psi.bAllConvergedInThisGSiteration = 1;
//		psi.iNRmax = 0;
//#pragma omp parallel 
//		{
//			int nrMax = 0;
//#pragma omp for schedule(guided) 
//			for (int i = 0; i < nCells; ++i) {
//				if (cvs[i].isSimulatingCell == 1) {
//					nrMax = calCEqUsingNRforCPU(i);
//					if (cvs[i].dp_tp1 > thdWet) {
//						setEffCells(i);
//					}
//				}
//			}
//#pragma omp critical(getMaxNR) 
//			{
//				if (nrMax > psi.iNRmax) {
//					psi.iNRmax = nrMax;
//				}
//			}
//		}
//		psi.iGSmax += 1;
//		if (psi.bAllConvergedInThisGSiteration == 1) {
//			break;
//		}
//	}
}
